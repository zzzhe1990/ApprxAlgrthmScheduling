#include "hip/hip_runtime.h"
#include "DPCUDA.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

int *dev_ATE_elm, *dev_ATE_myOPT, *dev_ATE_myOptimalindex, *dev_ATE_myMinNSVector;
	int *dev_ATE_NSsubsets, *dev_ATE_Csubsets;
	int *dev_ATE_optVector;
	int *dev_counterVec;
	int *dev_ATE_NSsubsets_size;
	int *dev_ATE_optVector_size;
	int *dev_zeroVec, *dev_roundVec;
	int *it, *ss, *NS;
	
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
	
void InitGPUData(int powK, int LongJobs_size, vector<DynamicTable> &AllTableElemets, int *zeroVec, 
				 int *roundVec, int *counterVec, int &maxSubsetsSize, const int maxSumValue, const int counterVecSize)
{
	cout << "At the beginning of InitGPUData. maxSubsetsSize: " << maxSubsetsSize << endl;
	
	int maxIndex = AllTableElemets.size() - 1;
	int maxCounterVec = 0;
	vector<int> temp;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}
	
	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}
	
	for (int i = 0; i < counterVecSize; i++)
	{
		if (counterVec[i] > maxCounterVec)
			maxCounterVec = counterVec[i];
	}
	
	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;
	
	//arrays on device
	gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets.size() * (maxSumValue +1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, (LongJobs_size + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&it, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&ss, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&NS, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));
    
	int *ATE_myOPT = new int[AllTableElemets.size()];
	for (int i = 0; i < AllTableElemets.size(); i++){
		gpuErrchk(hipMemcpy(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}
	
	gpuErrchk(hipMemcpy(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));
	
	delete(ATE_myOPT);
}


__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(const int *Ntemp, int *it, int Ntemp_size, int thread, int counter)
{
	
	printf("At the beginning of gpu_increase, thread: %d, counter: %d\n", thread, counter);
	
	for (int i = 0, size = Ntemp_size; i != size; ++i) 
	{
		const int index = size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} 
		else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, const int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, 
							  const int T, const int powK, int *it, int *s, int *NS, int *subsets_size, const int thread){
	
	//Ntemp_size = pow(k,2)
	
	//vector<int> it(Ntemp.size(), 0);
	//int it[Ntemp_size];
	int counter = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
	{
		it[i] = 0;
	}
		
//	if (thread == 0)
	{
		printf("In gpu_generate2 before do while loop, thread: %d.\n", thread);
	}
	
    do {
        //int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s, dev_roundVec, powK);
		
		
//        if(thread == 0)
        {
			printf("In gpu_generate2 in do while loop, counter: %d, thread: %d\n", counter, thread);
		}
			
        if(sSum <= T)
        {
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counter*Ntemp_size + i] = s[i];
			}
            //Ctemp.push_back(s);
            
//            if(thread == 0)
            {
				printf("Ctemp is updated successfully, counter: %d, thread: %d\n", counter, thread);
            }
            
            //int NS[Ntemp_size];
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
            
//            if(thread == 0)
            {
				printf("NS is updated successfully, counter: %d, thread: %d\n", counter, thread);
            }
            
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size)){
				
//				if(thread == 0)
				{
					printf("sameVector returns true, counter: %d, thread: %d\n", counter, thread);
				}
				
                continue;
			}
			
			
//			if(thread == 0)
			{
				printf("sameVector returns false, counter: %d, thread: %d\n", counter, thread);
			}
			
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counter * Ntemp_size + i] = NS[i];
			}
            //NMinusStemp.push_back(NS);
            
//            if(thread == 0)
            {
				printf("NMinusStemp is updated successfully, counter: %d, thread: %d\n", counter, thread);
			}
				
			counter++;
        }
    }while (gpu_increase(Ntemp, it, Ntemp_size, thread, counter));
    
    *subsets_size = counter;
    
    printf("At the end of gpu_generate2, thread: %d\n", thread);
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, 
						const int k, const int powK, const int AllTableElemets_size, int *dev_ATE_Csubsets, 
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec, 
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it, 
						int *s, int *NS, const int maxSubsetsSize){		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
		
		int j = thread + indexomp;
		if (thread < dev_counterVec[i]){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
            
//            if (thread == 0)
				printf("Before gpu_generate2, counterVec[%d]: %d, thread: %d\n", i, dev_counterVec[i], thread);
            gpu_generate2(&dev_ATE_elm[j * powK], powK, &dev_ATE_Csubsets[j * maxSubsetsSize * powK], &dev_ATE_NSsubsets[j * maxSubsetsSize * powK],
						  dev_roundVec, T, powK, &it[j * powK], &s[j * powK], &NS[j * powK], &dev_ATE_NSsubsets_size[j], thread);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
			
			__syncthreads();
			
//			if (thread == 0)
				printf("gpu_generate2 completed successfully. NSsubsets_size[%d]: %d, thread: %d\n", j, dev_ATE_NSsubsets_size[j], thread);
//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
//			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * powK + optVecIndex] = 0;
                    optVecIndex++;
                    dev_ATE_optVector_size[j] = optVecIndex;                                  
                    break;
                }             
                
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[j * powK], powK) ){
                    dev_ATE_optVector_size[j] = optVecIndex;

                    break;
				}
				
				
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < AllTableElemets_size; r++)
                {					
                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 
                    if (gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[r * powK], powK))
                    {
						
						if (thread == 0 && i == 19)
						{
							printf("level 3 starts. j: %d, optVecIndex: %d\n", j, optVecIndex);
						}
						
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * powK + optVecIndex] = dev_ATE_myOPT[r];
                        optVecIndex++;
						dev_ATE_optVector_size[j] = optVecIndex;
						
						
						if (thread == 0 && i == 19)
						{
							printf("level 3 complete. h: %d, r: %d\n", h, r);
						}			
				
                        break;
                    }
                }
            }
			
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
       //         cout << AllTableElemets[j].optVector[pp]<<" ";
//                if(AllTableElemets[j].optVector[pp] < minn)
				//if (thread == 0)
				//	printf("j: %d, thread: %d, AllTableElemets[%d].optVector[%d]: %d\n", j, thread, j, pp, dev_ATE_optVector[j*powK+pp]);
				if (dev_ATE_optVector[j * powK + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * powK + pp];
                    myOptimalindex=pp;
                }
            }
          //  cout << endl;
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
				for (int i = 0; i < powK; i++){
					dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * maxSubsetsSize + myOptimalindex) * powK + i];
				}
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }			   
		}//end if (j)
	}//end FindOPT()

void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, const int maxSumValue, 
			vector<int> &counterVec, const int LongJobs_size, int *zeroVec, int *roundVec)
{
    int ii=0;
    int indexomp=0;
    int maxSubsetsSize = 0;
	
	InitGPUData(powK, LongJobs_size, AllTableElemets, zeroVec, roundVec, &counterVec[0], maxSubsetsSize, maxSumValue, counterVec.size());

	cout << ", LongJob size: " << LongJobs_size << ", maxSumValue: " << maxSumValue << endl;
	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 32;
		int bSize = 1;
		if (tSize < counterVec[ii]){
			bSize = (tSize + counterVec[ii] - 1) / tSize;
		}
//		std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", indexomp: " << indexomp << std::endl;
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec, T, k, powK, 
								  AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets, 
								  dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_optVector, 
								  dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex, 
								  dev_ATE_myMinNSVector, ii, it, ss, NS, maxSubsetsSize);
           
//        gpuErrchk(hipMemcpy(&counterVec[0], dev_counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyDeviceToHost));
        indexomp+=counterVec[ii];
        ii++;
    } 
    
//GPU code to update AllTableElement
	int *temp_NSsubsets, *temp_Csubsets, *temp_myOPT, *temp_myOptIndex, *temp_myMinNSVector, *temp_optVector;
	temp_NSsubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_Csubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_myOPT = new int[AllTableElemets.size()];
	temp_myOptIndex = new int[AllTableElemets.size()];
	temp_myMinNSVector = new int[AllTableElemets.size() * powK];
	temp_optVector = new int[AllTableElemets.size() * (maxSumValue + 1)];
	
	cout << "FindOPT recursion is done. Start memcpy from Device to Host." << endl;
	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << endl;
	
	gpuErrchk(hipMemcpy(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myMinNSVector, dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_optVector, dev_ATE_optVector, AllTableElemets.size() * (maxSumValue + 1) * sizeof(int), hipMemcpyDeviceToHost));
	
	std::cout << "memcpy from device to host are done, AllTableElemets.size: " << AllTableElemets.size() << std::endl;

	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp_myOPT[i];
		AllTableElemets[i].myOptimalindex = temp_myOptIndex[i];
		int begin = 0, end = maxSubsetsSize * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp_NSsubsets[(i * maxSubsetsSize) * powK], &temp_NSsubsets[(i * maxSubsetsSize + 1) * powK]));
			AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp_Csubsets[(i * maxSubsetsSize) * powK], &temp_Csubsets[(i * maxSubsetsSize + 1) * powK]));
			begin += powK;
		}
		AllTableElemets[i].optVector.insert(AllTableElemets[i].optVector.end(), &temp_optVector[i * (maxSumValue + 1)], &temp_optVector[(i + 1) * (maxSumValue + 1)]);
		AllTableElemets[i].myMinNSVector.insert(AllTableElemets[i].myMinNSVector.end(), &temp_myMinNSVector[i * powK], &temp_myMinNSVector[(i + 1) * powK]);
	}
	
	
	free(temp_NSsubsets);
	free(temp_Csubsets);
	free(temp_myOPT);
	free(temp_myOptIndex);
	free(temp_myMinNSVector);
	free(temp_optVector);
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	hipFree(dev_ATE_optVector_size);
}
