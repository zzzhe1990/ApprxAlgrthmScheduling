#include "hip/hip_runtime.h"
#include "DPCUDA.h"

int *dev_ATE_elm, *dev_ATE_myOPT, *dev_ATE_myOptimalindex, *dev_ATE_myMinNSVector;
	int *dev_ATE_NSsubsets, *dev_ATE_Csubsets;
	int *dev_ATE_optVector;
	int *dev_counterVec;
	int *dev_ATE_NSsubsets_size;
	int *dev_ATE_optVector_size;
	int *dev_zeroVec, *dev_roundVec;
	int *it, *ss, *NS;
	
void InitGPUData(int AllTableElemets_size, int Cwhole_size, int powK, int LongJobs_size, 
				 vector<DynamicTable> &AllTableElemets, int *zeroVec, int *roundVec, int *counterVec)
{
	//arrays on device
	hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets_size * Cwhole_size * powK * sizeof(int));
	hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets_size * Cwhole_size * powK * sizeof(int));
	hipMalloc((void**)&dev_ATE_elm, AllTableElemets_size * powK * sizeof(int));
	hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets_size * powK * sizeof(int));
	hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets_size * sizeof(int));
	hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets_size * sizeof(int));
	hipMalloc((void**)&dev_ATE_optVector, AllTableElemets_size * powK * sizeof(int));
	hipMalloc((void**)&dev_counterVec, (LongJobs_size + 1) * sizeof(int));
	hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int));
	hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int));
	hipMalloc((void**)&it, powK * sizeof(int));
	hipMalloc((void**)&ss, powK * sizeof(int));
	hipMalloc((void**)&NS, powK * sizeof(int));
	hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets_size * sizeof(int));
    hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets_size * sizeof(int));
    
	int *ATE_NSsubsets_size = new int[AllTableElemets_size];
	int *ATE_optVector_size = new int[AllTableElemets_size];
	for (int i = 0; i < AllTableElemets_size; i++){
		ATE_NSsubsets_size[i] = AllTableElemets[i].NSsubsets.size();
		ATE_optVector_size[i] = AllTableElemets[i].optVector.size();
		hipMemcpy(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice);
	}
    
	hipMemcpy(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ATE_NSsubsets_size, ATE_NSsubsets_size, AllTableElemets_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ATE_optVector_size, ATE_optVector_size, AllTableElemets_size * sizeof(int), hipMemcpyHostToDevice);
	
	delete(ATE_NSsubsets_size);
	delete(ATE_optVector_size);
}


__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(const int *Ntemp, int *it, int Ntemp_size){
#pragma unroll
	for (int i = 0, size = Ntemp_size; i != size; ++i) {
		const int index = size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, const int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, 
							  const int T, const int powK, int *it, int *s, int *NS){
	
	//Ntemp_size = pow(k,2)
	
	//vector<int> it(Ntemp.size(), 0);
	//int it[Ntemp_size];
	int counter = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
		it[i] = 0;
    do {
        //int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s, dev_roundVec, powK);
        if(sSum <= T)
        {
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counter*Ntemp_size + i] = s[i];
			}
            //Ctemp.push_back(s);
            
            //int NS[Ntemp_size];
#pragma unroll
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size))
                continue;
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counter * Ntemp_size + i] = NS[i];
			}
            //NMinusStemp.push_back(NS);
        }
        counter++;
    }while (gpu_increase(Ntemp, it, Ntemp_size));
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, 
						const int k, const int powK, const int dev_AllTableElemets_size, int *dev_ATE_Csubsets, 
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int Cwhole_size, int *dev_zeroVec, 
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it, 
						int *s, int *NS){		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
			printf("thread: %d, i: %d, dev_counterVec[i]: %d\n", thread, i, dev_counterVec[i]);
		int j = thread + indexomp;
		if (thread < dev_counterVec[i]){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
            
            gpu_generate2(&dev_ATE_elm[j * powK], powK, dev_ATE_Csubsets, dev_ATE_NSsubsets, dev_roundVec, T, powK, it , s, NS);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * powK + optVecIndex] = 0;
                    optVecIndex++;
                    break;
                }
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], &dev_ATE_elm[j * powK], powK) )
                    break;
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < dev_AllTableElemets_size; r++)
                {

                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 
                    if (dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK] == dev_ATE_elm[r * powK])
                    {
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * powK + optVecIndex] = dev_ATE_myOPT[r];
                        optVecIndex++;
                        break;
                    }
                }
            }
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
#pragma unroll
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
       //         cout << AllTableElemets[j].optVector[pp]<<" ";
//                if(AllTableElemets[j].optVector[pp] < minn)
				if (dev_ATE_optVector[j * powK + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * powK + pp];
                    myOptimalindex=pp;
                }
            }
          //  cout << endl;
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
				for (int i = 0; i < powK; i++){
					dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * Cwhole_size + myOptimalindex) * powK + i];
				}
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }
		}//end if (j)
	}//end FindOPT()
/*
void gpu_DP(vector<DynamicTable> &AllTableElemets, int *dev_ATE_elm, int *dev_counterVec, int *dev_roundVec, 
			const int T, const int k, const int powK, const int dev_AllTableElemets_size,
			int *dev_ATE_Csubsets, int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, 
			int Cwhole_size, int *dev_zeroVec, int *dev_ATE_optVector, int *dev_ATE_optVector_size,
			int *dev_ATE_myOPT, int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, 
			int *it, int *s, int *NS, const int maxSumValue, vector<int> &counterVec)*/
void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, 
			const int dev_AllTableElemets_size, int Cwhole_size, const int maxSumValue, 
			vector<int> &counterVec)
{
    int ii=0;
    int indexomp=0;

	for (int i = 0; i < maxSumValue+1; i++){
		std::cout << "counterVec[" << i << "]: " << counterVec[i] << std::endl;
	}

	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 32;
		int bSize = 1;
		if (tSize < counterVec[ii]){
			bSize = (tSize + counterVec[ii] - 1) / tSize;
		}
		std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", bSize: " << bSize << ", tSize: " << tSize << std::endl;
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec, T, k, powK, 
								  dev_AllTableElemets_size, dev_ATE_Csubsets, dev_ATE_NSsubsets, 
								  dev_ATE_NSsubsets_size, Cwhole_size, dev_zeroVec, dev_ATE_optVector, 
								  dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex, 
								  dev_ATE_myMinNSVector, ii, it, ss, NS);
        
               
        hipMemcpy(&counterVec[0], dev_counterVec, powK * sizeof(int), hipMemcpyDeviceToHost);

        indexomp+=counterVec[ii];
        ii++;
    }
    
//GPU code to update AllTableElement
	for(int i=0; i < AllTableElemets.size(); i++){
		hipMemcpy(&AllTableElemets[i].NSsubsets[0][0], &dev_ATE_NSsubsets[i * Cwhole_size * powK], Cwhole_size * powK, hipMemcpyDeviceToHost);
		hipMemcpy(&AllTableElemets[i].optVector[0], &dev_ATE_optVector[i * powK], powK, hipMemcpyDeviceToHost);
		//Csubsets[Cwhole.size()][powK]
		hipMemcpy(&AllTableElemets[i].Csubsets[0][0], &dev_ATE_Csubsets[i * Cwhole_size * powK], Cwhole_size * powK, hipMemcpyDeviceToHost);
	}
	
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
}
