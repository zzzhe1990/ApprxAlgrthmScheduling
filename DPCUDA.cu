#include "hip/hip_runtime.h"
#include "DPCUDA.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


const int maxStreamNum = 32;
hipStream_t streams[maxStreamNum];

	
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

	
void InitGPUData(int powK, int LongJobs_size, vector<DynamicTable> &AllTableElemets, int *zeroVec, 
				 int *roundVec, int *counterVec, int &maxSubsetsSize, const int optVectorSize, 
				 const int counterVecSize, int **dev_ATE_elm, int **dev_ATE_myOPT, int **dev_ATE_myOptimalindex, 
				 int **dev_ATE_myMinNSVector, int **dev_ATE_NSsubsets, int **dev_ATE_Csubsets, int **dev_ATE_optVector,
				 int **dev_counterVec, int **dev_ATE_NSsubsets_size, int **dev_ATE_optVector_size, int **dev_zeroVec, 
				 int **dev_roundVec, int **it, int **ss, int **NS)
{	
	//cout << "Beginning of InitGPUData, thread: " << omp_get_thread_num() << ", dev_roundVec address: " << *dev_roundVec << endl;
	
	int maxIndex = AllTableElemets.size() - 1;
	int maxCounterVec = 0;
	vector<int> temp;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}
	
	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}
	
	for (int i = 0; i < counterVecSize; i++)
	{
		if (counterVec[i] > maxCounterVec)
			maxCounterVec = counterVec[i];
	}
	
	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;
	
	//arrays on device
	gpuErrchk(hipMalloc((void***)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_counterVec, (LongJobs_size + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&it, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&ss, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&NS, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void***)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));
    
    
	//cout << "thread: " << omp_get_thread_num() << ", cuda initial is done." << " Start memcpy! AllTableElemets.size: " << AllTableElemets.size() << endl;
    
    
	int *ATE_myOPT = new int[AllTableElemets.size()];
	
	for (int i = 0; i < AllTableElemets.size(); i++){
		gpuErrchk(hipMemcpyAsync(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice, 0));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}
	
	//gpuErrchk(hipMemcpyAsync(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(*dev_zeroVec, 0, powK*sizeof(int)));	
	gpuErrchk(hipMemcpy(*dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(*dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(*dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));
	
	delete[] ATE_myOPT;
	//cout << "End of InitGPUData, thread: " << omp_get_thread_num() << ", dev_roundVec address: " << *dev_roundVec << endl;
}


__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(int *Ntemp, int *it, int Ntemp_size)
{
	int index;
	for (int i = 0; i < Ntemp_size; i++) 
	{
		index = Ntemp_size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} 
		else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
//#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, const int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, 
							  const int T, const int powK, int *it, int *s, int *NS, int *subsets_size, const int thread){
	//vector<int> it(Ntemp.size(), 0);
	//int it[Ntemp_size];
	int counterNS = 0, counterC = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
	{
		it[i] = 0;
	}

    do {
        //int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s, dev_roundVec, powK);
			
        if(sSum <= T)
        {
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counterC*Ntemp_size + i] = s[i];
			}
			counterC++;
            //Ctemp.push_back(s);
            
            //int NS[Ntemp_size];
#pragma unroll
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
           
            
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size)){
                continue;
			}
#pragma unroll			
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counterNS * Ntemp_size + i] = NS[i];
			}
            //NMinusStemp.push_back(NS);
			
			counterNS++;
        }
    }while (gpu_increase(Ntemp, it, Ntemp_size));
    
    *subsets_size = counterNS;
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, 
						const int k, const int powK, const int AllTableElemets_size, int *dev_ATE_Csubsets, 
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec, 
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it, 
						int *s, int *NS, const int maxSubsetsSize, const int optVectorSize)
{		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
				
		int j = thread + indexomp;
		if (thread < dev_counterVec[i]){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
			
            gpu_generate2(&dev_ATE_elm[j * powK], powK, &dev_ATE_Csubsets[j * maxSubsetsSize * powK],
            			&dev_ATE_NSsubsets[j * maxSubsetsSize * powK], dev_roundVec, T, powK,
            			&it[thread * powK], &s[thread * powK], &NS[thread * powK],
            			&dev_ATE_NSsubsets_size[j], thread);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
			
			__syncthreads();
			

//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * optVectorSize + optVecIndex] = 0;
                    optVecIndex++;
                    dev_ATE_optVector_size[j] = optVecIndex;                                  
                    break;
                }             
                
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[j * powK], powK) ){
                    dev_ATE_optVector_size[j] = optVecIndex;

                    break;
				}
								
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < AllTableElemets_size; r++)
                {					
                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 

                   
                    if (gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[r * powK], powK))
                    {
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * optVectorSize + optVecIndex] = dev_ATE_myOPT[r];                       						
                        optVecIndex++;
						dev_ATE_optVector_size[j] = optVecIndex;
											
                        break;
                    }
                }
                
			}
		
			
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.         
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
				if (dev_ATE_optVector[j * optVectorSize + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * optVectorSize + pp];
                    myOptimalindex=pp;
                }
            }
			
				
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
				for (int i = 0; i < powK; i++){
					dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * maxSubsetsSize + myOptimalindex) * powK + i];
				}
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }			   
		}//end if (j)
		
		
}//end FindOPT()

void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, const int maxSumValue, 
			vector<int> &counterVec, const int LongJobs_size, int *zeroVec, int *roundVec)
{
	hipSetDevice(0);
	
	const int thread = omp_get_thread_num();
	const int numThreads = omp_get_num_threads();
	const int batchSize = maxStreamNum/numThreads;
	
	for (int i=0; i < batchSize; i++)
		hipStreamCreate(&streams[thread*batchSize+i]);
		
	int *dev_ATE_elm = 0, *dev_ATE_myOPT = 0, *dev_ATE_myOptimalindex = 0, *dev_ATE_myMinNSVector = 0;
	int *dev_ATE_NSsubsets = 0, *dev_ATE_Csubsets = 0;
	int *dev_ATE_optVector = 0;
	int *dev_counterVec = 0;
	int *dev_ATE_NSsubsets_size = 0;
	int *dev_ATE_optVector_size = 0;
	int *dev_zeroVec = 0, *dev_roundVec = 0;
	int *it = 0, *ss = 0, *NS = 0;
	
    int ii=0;
    int indexomp=0;
    int maxSubsetsSize = 0;
	const int optVectorSize = 64;
	
	
//	InitGPUData(powK, LongJobs_size, AllTableElemets, zeroVec, roundVec, &counterVec[0], maxSubsetsSize, optVectorSize, counterVec.size(),
//				&dev_ATE_elm, &dev_ATE_myOPT, &dev_ATE_myOptimalindex, &dev_ATE_myMinNSVector, &dev_ATE_NSsubsets, &dev_ATE_Csubsets,
//				&dev_ATE_optVector, &dev_counterVec, &dev_ATE_NSsubsets_size, &dev_ATE_optVector_size, &dev_zeroVec, &dev_roundVec, &it, &ss, &NS);	
	
	int maxIndex = AllTableElemets.size() - 1;
	int maxCounterVec = 0;
	vector<int> temp;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}
	
	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
//		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}
	
	maxSubsetsSize = 256;


	for (int i = 0; i < counterVec.size(); i++)
	{
		cout <<"thread: " << omp_get_thread_num() << ", counterVec[" << i << "]: " << counterVec[i] << endl;
		if (counterVec[i] > maxCounterVec)
			maxCounterVec = counterVec[i];
	}
	
//	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;
	
	//arrays on device
	gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, counterVec.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&it, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&ss, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&NS, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));
    
    
	//cout << "thread: " << omp_get_thread_num() << ", cuda initial is done." << " Start memcpy! AllTableElemets.size: " << AllTableElemets.size() << endl;
    
    
	int *ATE_myOPT = new int[AllTableElemets.size()];
	
	for (int i = 0; i < AllTableElemets.size(); i++){
		gpuErrchk(hipMemcpyAsync(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}
	
	//gpuErrchk(hipMemcpyAsync(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemsetAsync(dev_zeroVec, 0, powK*sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_counterVec, &counterVec[0], counterVec.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	
	
//	cout << ", LongJob size: " << LongJobs_size << ", maxSumValue: " << maxSumValue << endl;
    	
	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 256;
		int bSize = 1;

		for (int ptr=0; ptr < batchSize; ptr++)
		{
			if (ii < maxSumValue+1)
			{
				if (tSize < counterVec[ii]){
					bSize = (tSize + counterVec[ii] - 1) / tSize;
				}

				int sizeOffset = ptr * maxCounterVec * powK;
//		std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", indexomp: " << indexomp << std::endl;
				FindOPT<<<bSize, tSize, 0, streams[thread*batchSize+ptr]>>>(dev_ATE_elm,
									dev_counterVec, indexomp, dev_roundVec, T, k, powK,
									AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets,
									dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_optVector,
									dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex,
									dev_ATE_myMinNSVector, ii, &it[sizeOffset], &ss[sizeOffset],
									&NS[sizeOffset], maxSubsetsSize, optVectorSize);

				hipStreamSynchronize(streams[thread*batchSize+ptr]);

				indexomp+=counterVec[ii];
				ii++;
			}
		}
    } 
    
//	for (int i=0; i<batchSize; i++)
//	{
//		hipStreamSynchronize(streams[thread*batchSize+i]);
//	}

/*********************  GPU code to update AllTableElement  ********************************/
	int *temp_NSsubsets, *temp_Csubsets, *temp_myOPT, *temp_myOptIndex, *temp_myMinNSVector, *temp_optVector;
	temp_NSsubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_Csubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_myOPT = new int[AllTableElemets.size()];
	temp_myOptIndex = new int[AllTableElemets.size()];
	temp_myMinNSVector = new int[AllTableElemets.size() * powK];
	temp_optVector = new int[AllTableElemets.size() * optVectorSize];
	
//	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << endl;
	
	gpuErrchk(hipMemcpyAsync(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myMinNSVector, dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_optVector, dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	
	struct timeval t1, t2;
	gettimeofday(&t1, NULL);
	
	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp_myOPT[i];
		AllTableElemets[i].myOptimalindex = temp_myOptIndex[i];
		int begin = 0, end = maxSubsetsSize * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp_NSsubsets[(i * maxSubsetsSize) * powK], &temp_NSsubsets[(i * maxSubsetsSize + 1) * powK]));
			AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp_Csubsets[(i * maxSubsetsSize) * powK], &temp_Csubsets[(i * maxSubsetsSize + 1) * powK]));
			begin += powK;
		}
		AllTableElemets[i].optVector.insert(AllTableElemets[i].optVector.end(), &temp_optVector[i * optVectorSize], &temp_optVector[(i + 1) * optVectorSize]);
		AllTableElemets[i].myMinNSVector.insert(AllTableElemets[i].myMinNSVector.end(), &temp_myMinNSVector[i * powK], &temp_myMinNSVector[(i + 1) * powK]);
	}
	
	gettimeofday(&t2, NULL);
	cout << "memory transfer to vectors Runtime: "  << t2.tv_sec - t1.tv_sec << endl;

	
	delete[] ATE_myOPT;
	delete[] temp_NSsubsets;
	delete[] temp_Csubsets;
	delete[] temp_myOPT;
	delete[] temp_myOptIndex;
	delete[] temp_myMinNSVector;
	delete[] temp_optVector;
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	hipFree(dev_ATE_optVector_size);

	for (int i=0; i<batchSize; i++)
		hipStreamDestroy(streams[thread*batchSize+i]);
}
