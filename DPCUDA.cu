#include "hip/hip_runtime.h"
#include "DPCUDA.h"
#include "sameVector.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


const int maxStreamNum = 64;
hipStream_t streams[maxStreamNum];


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
void InitGPUData(int powK, int LongJobs_size, vector<DynamicTable> &AllTableElemets, int *zeroVec,
				 int *roundVec, int *counterVec, int &maxSubsetsSize, const int optVectorSize,
				 const int counterVecSize, int **dev_ATE_elm, int **dev_ATE_myOPT, int **dev_ATE_myOptimalindex,
				 int **dev_ATE_myMinNSVector, int **dev_ATE_NSsubsets, int **dev_ATE_Csubsets, int **dev_ATE_optVector,
				 int **dev_counterVec, int **dev_ATE_NSsubsets_size, int **dev_ATE_optVector_size, int **dev_zeroVec,
				 int **dev_roundVec, int **it, int **ss, int **NS)
{
	//cout << "Beginning of InitGPUData, thread: " << omp_get_thread_num() << ", dev_roundVec address: " << *dev_roundVec << endl;

	int maxIndex = AllTableElemets.size() - 1;
	int maxCounterVec = 0;
	vector<int> temp;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}

	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}

	for (int i = 0; i < counterVecSize; i++)
	{
		if (counterVec[i] > maxCounterVec)
			maxCounterVec = counterVec[i];
	}

	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;

	//arrays on device
	gpuErrchk(hipMalloc((void***)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_counterVec, (LongJobs_size + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&it, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&ss, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&NS, maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void***)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void***)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));


	//cout << "thread: " << omp_get_thread_num() << ", cuda initial is done." << " Start memcpy! AllTableElemets.size: " << AllTableElemets.size() << endl;


	int *ATE_myOPT = new int[AllTableElemets.size()];

	for (int i = 0; i < AllTableElemets.size(); i++){
		gpuErrchk(hipMemcpyAsync(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice, 0));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}

	//gpuErrchk(hipMemcpyAsync(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(*dev_zeroVec, 0, powK*sizeof(int)));
	gpuErrchk(hipMemcpy(*dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(*dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(*dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));

	delete[] ATE_myOPT;
	//cout << "End of InitGPUData, thread: " << omp_get_thread_num() << ", dev_roundVec address: " << *dev_roundVec << endl;
}
*/
/*
__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}
	}
	return same;
}

__device__ int gpu_sameVectors(int *vecA, int choice, int size)
{
	int vecB[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0};

	vecB[15] = choice;

	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}
	}
	return same;
}


__global__ void gpu_sameVectors(int *A, int *B, const int powK, int *res)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tRes;
//	__shared__ int warpRes[32];

//	if (thread < 32)
//		warpRes[thread] = 0;

	if (thread < powK)
	{
		tRes = __all(A[thread]-B[thread]);
	}

//	if (thread&(32-1) == 0)
//	{
//		warpRes[thread/32] = tRes;
//	}

//	if (thread < 32)
//		tRes = __any( warpRes != 0 );

	if (thread == 0)
		res[0] = tRes;
}
*/

__device__ int gpu_increase(int *Ntemp, int *it, int Ntemp_size)
{
	int index;
	for (int i = 0; i < Ntemp_size; i++)
	{
		index = Ntemp_size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		}
		else {
			return 1;
		}
	}
	return 0;
}


__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
//#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__host__ __device__ int gpu_blockOffsetNoZero(int *block, int *div, int blockSize, int divSize)
{
	int blockOffset = 0;
	for (int i = 0; i < blockSize; i++)
	{
		int offset = block[i];
		for (int j = i+1 ; j < divSize; j++)
			offset *= div[j];
		blockOffset += offset;
	}
	return blockOffset;
}

__device__ int gpu_blockOffset(int *block, int *divComp, int blockSize, int *div, int divSize)
{
	int blockOffset = 0, count = 0;
	for (int i = 0; i < blockSize; i++)
	{
		if (divComp[i] != 0)
		{
			int offset = block[i];
			for (int j = count+1 ; j < divSize; j++)
				offset *= div[j];
			blockOffset += offset;
			count++;
		}
	}
	return blockOffset;
}


//FOr the selecting current configure Nsub, find all its sub-configures and update corresponding C set and NSsub set
__global__ void gpu_genSubConfigs(int *jNSub, int *jC, int *jNSsubsets, int *jConfigSize, const int jNSize,
								  const int totalThread, const int powK, const int T, int *jCountC,
								  int *jCountNS, int *dev_roundVec)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;

	//this is the maximum # of dimensions that a configuration can have

	if (thread < totalThread)
	{
		int subConfig[64];
		int NS[64];
		int remain = thread;
		int index = 0;
		//For a given thread, find its corresponding sub-configure and store into subConfig at first jNsize pos.
		for (int i=0; i < powK; i++)
		{
			if (jNSub[i] == 0)
			{
				subConfig[i] = 0;
			}
			else
			{
				int offset = 1;
				for (int j = index+1; j < jNSize; j++)
					offset *= jConfigSize[j];

				subConfig[i] = remain / offset;
				remain -= (subConfig[i] * offset);
				index++;
			}
		}

		int sSum=gpu_sumFun(&subConfig[0], dev_roundVec, powK);

		if(sSum <= T)
		{
			jCountC[thread] = 1;
			for (int i = 0; i < powK; i++)
			{
				jC[thread*powK + i] = subConfig[i];
				NS[i] = jNSub[i] - subConfig[i];
			}

			if(gpu_sameVectors(&NS[0], jNSub, powK) == 0)
			{
				jCountNS[thread] = 1;
				for (int i = 0; i < powK; i++)
				{
					jNSsubsets[thread * powK + i] = NS[i];
				}
			}
		}
	}
}

/*
__global__ void gpu_generate2(const int maxSubsetsSize, int *dev_ATE_elm, const int powK,
							  int *dev_ATE_Csubsets, int *dev_ATE_NSsubsets, int *dev_roundVec,
							  const int T, int *dev_ATE_NSsubsets_size,
							  int *dev_counterVec, const int ii, const int indexomp,
							  int *dev_subConfigSize, int *dev_countC, int *dev_countNS)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int j = thread + indexomp;

	if (thread < dev_counterVec[ii]){
		int *jN = &dev_ATE_elm[j * powK];
		int *jC = &dev_ATE_Csubsets[j * maxSubsetsSize * powK];
		int *jNSsubsets = &dev_ATE_NSsubsets[j * maxSubsetsSize * powK];
		int *jCountC = &dev_countC[j * maxSubsetsSize];
		int *jCountNS = &dev_countNS[j * maxSubsetsSize];
		int *jConfigSize = &dev_subConfigSize[thread * powK];

		int jNSize = 0;
		int numThread = 1;

		for (int i=0; i<powK; i++)
		{
			if (jN[i] != 0)
			{
				jConfigSize[jNSize] = jN[i] + 1;
				jNSize++;
				numThread *= (jN[i]+1);
			}
		}
		//hipDeviceSynchronize();

		int threadsPerBlock = 64;
		int blocksPerGrid = (numThread + threadsPerBlock -1) / threadsPerBlock;

		gpu_genSubConfigs<<<blocksPerGrid, threadsPerBlock>>>(jN, jC, jNSsubsets, jConfigSize, jNSize,
															numThread, powK, T, jCountC, jCountNS, dev_roundVec);
	}
}
*/

__global__ void FindAllSub(const int powK, int *jN, int *dev_roundVec, const int T, int *jNSsubsets, 
						   int *dev_lock, int *dev_ATE_NSsubsets_size, const int id, const int allSubSize,
						   int cpuId, int blockLvl, int blockIDInLvl)
{
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread < allSubSize){		
		int js[16];
		int jNS[16];
		
		int residual = thread;
		for (int i = 0; i < powK; i++)
		{
			js[i] = 0;
			int div = 1;
			if (jN[i] != 0){				
				for (int j=i+1; j<powK; j++){
					if (jN[j] != 0){
						div *= (jN[j] + 1);
					}
				}
				js[i] = residual / div;
				residual -= (div * js[i]);
			}
		}	

		int sSum=gpu_sumFun(js, dev_roundVec, powK);

		if(sSum <= T)
		{		
			for(int i=0; i<powK; i++)
			{
				jNS[i] = jN[i] - js[i];
			}
			
			if(gpu_sameVectors(jNS, jN, powK) == 0){	
				bool leave = true;
				while (leave) {
					if (atomicCAS(&dev_lock[id], 0, 1) == 0) {
						for (int i = 0; i < powK; i++)
						{
							jNSsubsets[dev_ATE_NSsubsets_size[id] * powK + i] = jNS[i];
						}
						atomicAdd(&dev_ATE_NSsubsets_size[id], 1);
						leave = false;
						
						atomicExch(&dev_lock[id], 0);
					}
				}
			}
		}
	}
}

/*
__global__ void gpu_generate2(const int maxSubsetsSize, int *dev_ATE_elm, const int powK,
							  int *dev_ATE_Csubsets, int *dev_ATE_NSsubsets, int *dev_roundVec,
							  const int T, int *it, int *s, int *NS, int *dev_ATE_NSsubsets_size,
							  int *dev_counterVec, const int ii, const int indexomp, int *dev_lock,
							  int cpuId, int blockLvl, int blockIDInLvl)
*/
__global__ void gpu_generate2(const int maxSubsetsSize, int *dev_ATE_elm, const int powK,
							  int *dev_ATE_NSsubsets, int *dev_roundVec, const int T, 
							  int *dev_ATE_NSsubsets_size, int *dev_counterVec, 
							  const int ii, const int indexomp, int *dev_lock, int cpuId, 
							  int blockLvl, int blockIDInLvl)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int j = thread + indexomp;

	if (thread < dev_counterVec[ii]){
		int *jN = &dev_ATE_elm[j * powK];
		//int *jC = &dev_ATE_Csubsets[j * maxSubsetsSize * powK];
		int *jNSsubsets = &dev_ATE_NSsubsets[j * maxSubsetsSize * powK];
		//int *jit = &it[thread * powK];
		//int *js = &s[thread * powK];
		//int *jNS = &NS[thread * powK];

		//int counterNS = 0, counterC = 0;
		int allSubSize = 1;

		for (int i = 0; i < powK; i++)
		{
			//jit[i] = 0;
			//js[i] = 0;
			//jNS[i] = 0;
			allSubSize *= (jN[i]+1);
		}
/*
//		if (allSubSize < 16){
			do {
				for (int i = 0; i < powK; i++)
				{
					js[i] = jit[i];
				}
									
				int sSum=gpu_sumFun(js, dev_roundVec, powK);

				if(sSum <= T)
				{
					for (int i = 0; i < powK; i++)
					{
						jC[counterC*powK + i] = js[i];
					}
										
					counterC++;		
					
					for(int i=0; i<powK; i++)
					{
						jNS[i] = jN[i] - js[i];
					}

					if(gpu_sameVectors(jNS, jN, powK)){
						continue;
					}

					for (int i = 0; i < powK; i++)
					{
						jNSsubsets[counterNS * powK + i] = jNS[i];
					}

					counterNS++;
				}
			}while (gpu_increase(jN, jit, powK));	
			dev_ATE_NSsubsets_size[j] = counterNS;
/*		}
		else{
*/			int blockSize = 32;
			int gridSize = (allSubSize + blockSize - 1) / blockSize;
			FindAllSub<<<gridSize, blockSize>>>(powK, jN, dev_roundVec, T, jNSsubsets, dev_lock, dev_ATE_NSsubsets_size, j, allSubSize, cpuId, blockLvl, blockIDInLvl);
//		}*/
	}
}

__global__ void FindSubConfigOPT(int MemOffset, int *NS, int *dev_ATE_elm, int powK, 
								 int *dev_ATE_optVector, int optOffset, int optVecIndex, 
								 int *dev_ATE_optVector_size, int j, int *dev_ATE_myOPT,
								 int jobsPerBlock)
{
	__shared__ int lock[1];
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	int r = MemOffset + thread;
	if (thread == 0)
		lock[1] = 0;
	__syncthreads();
	
	if (thread < jobsPerBlock){
		if (lock[1] == 0){
			if (gpu_sameVectors(NS, &dev_ATE_elm[r * powK], powK))
			{
				dev_ATE_optVector[optOffset + optVecIndex] = dev_ATE_myOPT[r];
				dev_ATE_optVector_size[j] = optVecIndex+1;
				atomicAdd(lock, 1);
			}
		}
	}
}

/*
__global__ void LoopNSsubsets(int *dev_ATE_NSsubsets_size, int *dev_ATE_NSsubsets, int NSOffset,
							  int powK, int *dev_zeroVec, int *blockDimSize, int *divisorComp, int *divisor, 
							  int divSize, int jobsPerBlock, int *dev_ATE_elm, volatile int *dev_ATE_myOPT,
							  const int j, const int cpuId, const int configOffset, int *dev_ATE_myMinNSVector,
							  int *dev_ATE_myOptimalindex, const int optOffset, int *dev_ATE_optVector)
*/
__global__ void LoopNSsubsets(int *dev_ATE_NSsubsets_size, int *dev_ATE_NSsubsets, int NSOffset,
							  int powK, int *dev_zeroVec, int *blockDimSize, int *divisorComp, int *divisor, 
							  int divSize, int jobsPerBlock, int *dev_ATE_elm, volatile int *dev_ATE_myOPT,
							  const int j, const int cpuId, const int configOffset, int *dev_ATE_myMinNSVector,
							  int *dev_ATE_myOptimalindex)
{
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	int *NS = &dev_ATE_NSsubsets[(NSOffset + thread) * powK];
		
	__shared__ int lock[1];
	__shared__ int opt[64];
	__shared__ int idx[64];
	
	//dev_ATE_optVector[optOffset + thread] = 1000000;
	opt[thread] = 100000;
	idx[thread] = thread;
	
	if (thread == 0)
		lock[0] = 0;
	__syncthreads();
	
	if (thread < dev_ATE_NSsubsets_size[j]){
		
		if(gpu_sameVectors(NS, dev_zeroVec, powK))
		{
			//dev_ATE_optVector[optOffset + thread] = 0;
			opt[thread] = 0;
			lock[0] = 1;
		}
	}
	__syncthreads();
	
	if (thread < dev_ATE_NSsubsets_size[j]){
		int blockIndex[32];
		
		if (lock[0] == 0){
			//Find in which block the sub-configuration is stored.
			for (int i = 0; i < powK; i++)
			{
				if (blockDimSize[i] != 0)
				{
					blockIndex[i] = NS[i] / blockDimSize[i];
				}
			}

			int blockOffset = gpu_blockOffset(&blockIndex[0], divisorComp, powK, divisor, divSize);
			int MemOffset = blockOffset * jobsPerBlock;
		
			for (int r = MemOffset; r < MemOffset + jobsPerBlock; r++)
			{	
				if (gpu_sameVectors(NS, &dev_ATE_elm[r * powK], powK))
				{
					//dev_ATE_optVector[optOffset + thread] = dev_ATE_myOPT[r];
					opt[thread] = dev_ATE_myOPT[r];
					break;
				}
			} 
		}
	}
	__syncthreads();
			
	if (thread < dev_ATE_NSsubsets_size[j]){
/*		if (thread > 32){
			if (dev_ATE_optVector[optOffset + thread] < dev_ATE_optVector[optOffset + thread - 32]){
				dev_ATE_optVector[optOffset + thread - 32] = dev_ATE_optVector[optOffset + thread];
				idx[thread-32] = idx[thread];
			}
		}
		__syncthreads();
		if (thread > 16 && thread <= 32 ){
			if (dev_ATE_optVector[optOffset + thread] < dev_ATE_optVector[optOffset + thread - 16]){
				dev_ATE_optVector[optOffset + thread - 16] = dev_ATE_optVector[optOffset + thread];
				idx[thread-16] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread > 8 && thread <= 16 ){
			if (dev_ATE_optVector[optOffset + thread] < dev_ATE_optVector[optOffset + thread - 8]){
				dev_ATE_optVector[optOffset + thread - 8] = dev_ATE_optVector[optOffset + thread];
				idx[thread-8] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread > 4 && thread <= 8 ){
			if (dev_ATE_optVector[optOffset + thread] < dev_ATE_optVector[optOffset + thread - 4]){
				dev_ATE_optVector[optOffset + thread - 4] = dev_ATE_optVector[optOffset + thread];
				idx[thread-4] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread > 2 && thread <= 4 ){
			if (dev_ATE_optVector[optOffset + thread] < dev_ATE_optVector[optOffset + thread - 2]){
				dev_ATE_optVector[optOffset + thread - 2] = dev_ATE_optVector[optOffset + thread];
				idx[thread-2] = idx[thread];
			}
		}			
		__syncthreads();
		
		if (thread == 0){
			if (dev_ATE_optVector[optOffset + 1] < dev_ATE_optVector[optOffset]){
				dev_ATE_optVector[optOffset] = dev_ATE_optVector[optOffset + 1];
				idx[0] = idx[1];
			}
			
			dev_ATE_myOPT[configOffset + j] = dev_ATE_optVector[optOffset] + 1;
			dev_ATE_myOptimalindex[configOffset + j] = idx[0];
		}
		__syncthreads();
*/		
		
		if (thread >= 32){
			if (opt[thread] < opt[thread - 32]){
				opt[thread - 32] = opt[thread];
				idx[thread-32] = idx[thread];
			}
		}
		__syncthreads();
		if (thread >= 16 && thread < 32 ){
			if (opt[thread] < opt[thread - 16]){
				opt[thread - 16] = opt[thread];
				idx[thread-16] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread >= 8 && thread < 16 ){
			if (opt[thread] < opt[thread - 8]){
				opt[thread - 8] = opt[thread];
				idx[thread-8] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread >= 4 && thread < 8 ){
			if (opt[thread] < opt[thread - 4]){
				opt[thread - 4] = opt[thread];
				idx[thread-4] = idx[thread];
			}
		}			
		__syncthreads();
		if (thread >= 2 && thread < 4 ){
			if (opt[thread] < opt[thread - 2]){
				opt[thread - 2] = opt[thread];
				idx[thread-2] = idx[thread];
			}
		}			
		__syncthreads();
		
		if (thread == 0){
			if (opt[1] < opt[0]){
				opt[0] = opt[1];
				idx[0] = idx[1];
			}
			
			dev_ATE_myOPT[configOffset + j] = opt[0] + 1;
			dev_ATE_myOptimalindex[configOffset + j] = idx[0];
		}
		__syncthreads();
	
		if (thread < powK){
			dev_ATE_myMinNSVector[(j + configOffset) * powK + thread] = dev_ATE_NSsubsets[(NSOffset + idx[0]) * powK + thread];
		}
	}
}

#ifdef SPLIT
/*
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec,
						const int powK, const int AllTableElemets_size, int *dev_ATE_Csubsets,
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec,
						int *dev_ATE_myOPT, int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, 
						const int ii, const int maxSubsetsSize, int *blockDimSize, int *divisor, 
						int *divisorComp, const int divSize, const int jobsPerBlock, const int cpuId, 
						const int configOffset, const int blockLvl, const int blockIDInLvl,
						int *dev_ATE_optVector, const int optVectorSize)
*/
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec,
						const int powK, const int AllTableElemets_size, int *dev_ATE_NSsubsets, 
						int *dev_ATE_NSsubsets_size, int *dev_zeroVec, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int ii, 
						const int maxSubsetsSize, int *blockDimSize, int *divisor, int *divisorComp, 
						const int divSize, const int jobsPerBlock, const int cpuId, const int configOffset, 
						const int blockLvl, const int blockIDInLvl)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;

	int j = thread + indexomp;
	int NSOffset = j * maxSubsetsSize;
	//int optOffset = j * optVectorSize;

	if (thread < dev_counterVec[ii]){
		
		int blockSize = 64;
		int gridSize = (dev_ATE_NSsubsets_size[j] + blockSize - 1) / blockSize;
/*				
		LoopNSsubsets<<<gridSize, blockSize>>>(dev_ATE_NSsubsets_size, dev_ATE_NSsubsets, NSOffset, 
												powK, dev_zeroVec, blockDimSize, divisorComp, divisor, 
												divSize, jobsPerBlock, dev_ATE_elm, dev_ATE_myOPT, j, 
												cpuId, configOffset, dev_ATE_myMinNSVector, 
												dev_ATE_myOptimalindex, optOffset, dev_ATE_optVector);
*/
		LoopNSsubsets<<<gridSize, blockSize>>>(dev_ATE_NSsubsets_size, dev_ATE_NSsubsets, NSOffset, 
												powK, dev_zeroVec, blockDimSize, divisorComp, divisor, 
												divSize, jobsPerBlock, dev_ATE_elm, dev_ATE_myOPT, j, 
												cpuId, configOffset, dev_ATE_myMinNSVector, dev_ATE_myOptimalindex);
	}
/*
		int blockIndex[64];
		for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
		{
			//actually, h == optVectorIndex, and optVector_size = optVecIndex+1;			
			int *NS = &dev_ATE_NSsubsets[(NSOffset + h) * powK];
			if(gpu_sameVectors(NS, dev_zeroVec, powK))
			{
				dev_ATE_optVector[optOffset + h] = 0;
				dev_ATE_optVector_size[j] = h+1;
				break;
			}

			//Find in which block the sub-configuration is stored.
			for (int i = 0; i < powK; i++)
			{
				if (blockDimSize[i] != 0)
				{
					blockIndex[i] = NS[i] / blockDimSize[i];
				}
			}

			int blockOffset = gpu_blockOffset(&blockIndex[0], divisorComp, powK, divisor, divSize);
			int MemOffset = blockOffset * jobsPerBlock;
			bool nomatch = true;
			for (int r = MemOffset; r < MemOffset + jobsPerBlock && nomatch; r++)
			{	
				if (gpu_sameVectors(NS, &dev_ATE_elm[r * powK], powK))
				{
					dev_ATE_optVector[optOffset + h] = dev_ATE_myOPT[r];
					dev_ATE_optVector_size[j] = h+1;
					nomatch = false;
				}
			} 
		}

	if (thread < dev_counterVec[ii]){
		int minn = 100000;
		int myOptimalindex = 0;
		// find out the OPT from all dependencies.		
		for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
		{			
			if (dev_ATE_optVector[optOffset + pp] < minn)
			{
				minn = dev_ATE_optVector[optOffset + pp];
				myOptimalindex=pp;
			}
		}
		
		int optTemp=minn+1;
		dev_ATE_myOPT[configOffset + j] = optTemp;
		dev_ATE_myOptimalindex[configOffset + j] = myOptimalindex;

		if (dev_ATE_NSsubsets_size[j] > 0)
		{
			for (int i = 0; i < powK; i++){
				dev_ATE_myMinNSVector[(j + configOffset) * powK + i] = dev_ATE_NSsubsets[(NSOffset + myOptimalindex) * powK + i];
			}
		}
	}//end if (j)
*/
}//end FindOPT()

#else

__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T,
						const int k, const int powK, const int AllTableElemets_size, int *dev_ATE_Csubsets,
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec,
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT,
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it,
						int *s, int *NS, const int maxSubsetsSize, const int optVectorSize)
{
	//something new and try
	int maxIndex;

	int thread = blockDim.x * blockIdx.x + threadIdx.x;

	int j = thread + indexomp;
	if (thread < dev_counterVec[i]){
//      for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j],
																// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job

		int optVecIndex = 0;

		for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
		{
			if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], dev_zeroVec, powK))
			{
				dev_ATE_optVector[j * optVectorSize + optVecIndex] = 0;
				optVecIndex++;
				dev_ATE_optVector_size[j] = optVecIndex;
				break;
			}

			for (int r = 0; r < AllTableElemets_size; r++)
			{
				if (gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[r * powK], powK))
				{
					//AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
					dev_ATE_optVector[j * optVectorSize + optVecIndex] = dev_ATE_myOPT[r];
					optVecIndex++;
					dev_ATE_optVector_size[j] = optVecIndex;

					break;
				}
			}
		}

		int minn = 100000;
		int myOptimalindex;
		//for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
		for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
		{
			if (dev_ATE_optVector[j * optVectorSize + pp] < minn)
			{
//                    minn=AllTableElemets[j].optVector[pp];
				minn = dev_ATE_optVector[j * optVectorSize + pp];
				myOptimalindex=pp;
			}
		}


		int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
		dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
		dev_ATE_myOptimalindex[j] = myOptimalindex;

//            if(AllTableElemets[j].NSsubsets.size()>0)
		if (dev_ATE_NSsubsets_size[j] > 0)
		{
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
			for (int i = 0; i < powK; i++){
				dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * maxSubsetsSize + myOptimalindex) * powK + i];
			}
			//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
		}
	}//end if (j)


}//end FindOPT()
#endif

/*
__global__ void LaunchBlocks(int blockOffset, int *dev_ATE_elm, int *dev_counterVec, int *dev_roundVec,
							const int powK, const int cpuId, const int AllTableElemets_size, int *dev_ATE_Csubsets,
							int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec, int *dev_ATE_myOPT,
							int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int blockLvl,
							const int maxSubsetsSize, int *dev_ATE_optVector, int jobsPerBlock,
							int *blockDimSize, const int configOffset, const int T, int *it, int *ss,
							int* NS, int *divisor, int *divisorComp, const int divSize, const int levelsPerBlock,
							const int blockIDInLvl, int *dev_lock1, const int optVectorSize)
*/
__global__ void LaunchBlocks(int blockOffset, int *dev_ATE_elm, int *dev_counterVec, int *dev_roundVec,
							const int powK, const int cpuId, const int AllTableElemets_size, 
							int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec, int *dev_ATE_myOPT,
							int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int blockLvl,
							const int maxSubsetsSize, int jobsPerBlock, int *blockDimSize, const int configOffset, 
							const int T, int *divisor, int *divisorComp, const int divSize, 
							const int levelsPerBlock, const int blockIDInLvl, int *dev_lock1)
{
	int ii = 0;
	int indexomp = 0;
	int *bN = &dev_ATE_elm[configOffset * powK];
	//int *bC = &dev_ATE_Csubsets[configOffset * maxSubsetsSize * powK];
	int *bNS = &dev_ATE_NSsubsets[configOffset * maxSubsetsSize * powK];
	int *bNS_size = &dev_ATE_NSsubsets_size[configOffset];
	//int *bOptVec = &dev_ATE_optVector[configOffset * optVectorSize];
	//int *bOptVec_size = &dev_ATE_optVector_size[configOffset];

	while (ii < levelsPerBlock)		//number of levels = the sum of each block dimension size -1
	{
		int tSize = 64;
		int bSize = 1;

		if (tSize < dev_counterVec[ii]){
			bSize = (tSize + dev_counterVec[ii] - 1) / tSize;
		}
/*
		gpu_generate2<<<bSize, tSize>>>(maxSubsetsSize, bN, powK, bC, bNS, dev_roundVec,
					T, it, ss, NS, bNS_size, dev_counterVec, ii, indexomp, dev_lock1, cpuId,
					blockLvl, blockIDInLvl);
*/					
		gpu_generate2<<<bSize, tSize>>>(maxSubsetsSize, bN, powK, bNS, dev_roundVec, T, 
										bNS_size, dev_counterVec, ii, indexomp, dev_lock1, 
										cpuId, blockLvl, blockIDInLvl);

		hipDeviceSynchronize();
/*
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec,
								powK, AllTableElemets_size, bC, bNS, bNS_size, dev_zeroVec,
								dev_ATE_myOPT, dev_ATE_myOptimalindex, dev_ATE_myMinNSVector, 
								ii, maxSubsetsSize, blockDimSize, divisor, divisorComp, divSize, 
								jobsPerBlock, cpuId, configOffset, blockLvl, blockIDInLvl,
								bOptVec, optVectorSize);
*/
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec,
								powK, AllTableElemets_size, bNS, bNS_size, dev_zeroVec,
								dev_ATE_myOPT, dev_ATE_myOptimalindex, dev_ATE_myMinNSVector, 
								ii, maxSubsetsSize, blockDimSize, divisor, divisorComp, divSize, 
								jobsPerBlock, cpuId, configOffset, blockLvl, blockIDInLvl);

		indexomp+=dev_counterVec[ii];
		ii++;
		hipDeviceSynchronize();
	}
}





#ifdef SPLIT
//****************************************************************************************************************
//*********************** This is function for multiple blocks ***************************************************
//****************************************************************************************************************
void gpu_BlockDP(vector<DynamicTable> &AllTableElemets, const int T, const int powK, const int jobsPerBlock,
			const int levelsPerBlock, vector<int> &counterVec, const int LongJobs_size, int *zeroVec,
			int *roundVec, vector<int> &divisor, vector<int> &divisorComp, vector<int> &blockDimSize,
			vector<block> &allBlocks, vector<block> &allBlocksNoZero, vector<int> &blockCounterVec)
{
	hipSetDevice(0);

	const int thread = omp_get_thread_num();
	const int numThreads = omp_get_num_threads();
	const int batchSize = maxStreamNum/numThreads;

	for (int i=0; i < batchSize; i++)
		hipStreamCreate(&streams[thread*batchSize+i]);

	int *dev_ATE_elm = 0, *dev_ATE_myOPT = 0, *dev_ATE_myOptimalindex = 0, *dev_ATE_myMinNSVector = 0;
	int *dev_ATE_NSsubsets = 0;
	//int *dev_ATE_Csubsets = 0;
	//int *dev_ATE_optVector = 0;
	int *dev_counterVec = 0;
	int *dev_ATE_NSsubsets_size = 0;
	//int *dev_ATE_optVector_size = 0;
	int *dev_zeroVec = 0, *dev_roundVec = 0;
	//int *it = 0, *ss = 0, *NS = 0;
	int *dev_blockDimSize, *dev_divisor, *dev_divisorComp;
	int *dev_lock1;//, *dev_lock2;
	//int *dev_ifSame;
	
    int ii=0;
    int indexomp=0;
    const int maxSubsetsSize = 64;
    //const int optVectorSize = 64;

/*
	int maxCounterVec = 0;
	int maxIndex = AllTableElemets.size() - 1;
	vector<int> temp;
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}

	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
//		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}
*/
	int maxBlockLvlSize = 0;
	int maxInBlockLvlSize = 0;

	for (int i = 0; i < blockCounterVec.size(); i++)
	{
		if (blockCounterVec[i] > maxBlockLvlSize)
			maxBlockLvlSize = blockCounterVec[i];
	}

	for (int i = 0; i < counterVec.size(); i++)
	{
		if (counterVec[i] > maxInBlockLvlSize)
			maxInBlockLvlSize = counterVec[i];
	}

#ifdef _DEVICE_DEBUG
	cout << "thread: " << thread << ", maxBlockLvlSize: " << maxBlockLvlSize << endl;
	cout << "thread: " << thread << ", maxInBlockLvlSize: " << maxInBlockLvlSize << endl;
	cout << "thread: " << thread << ", AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << endl;
#endif


	//arrays on device
	//gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, counterVec.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&it, batchSize * maxBlockLvlSize * maxInBlockLvlSize * powK * sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&ss, batchSize * maxBlockLvlSize * maxInBlockLvlSize * powK * sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&NS, batchSize * maxBlockLvlSize * maxInBlockLvlSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    //gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_blockDimSize, blockDimSize.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_divisor, divisor.size() * sizeof(int)))
    gpuErrchk(hipMalloc((void**)&dev_divisorComp, powK * sizeof(int)))
    gpuErrchk(hipMalloc((void**)&dev_lock1, AllTableElemets.size() * sizeof(int)));
    //gpuErrchk(hipMalloc((void**)&dev_lock2, AllTableElemets.size() * sizeof(int)));
    //gpuErrchk(hipMalloc((void**)&dev_ifSame, AllTableElemets.size() * sizeof(int)));
	//cout << "thread: " << omp_get_thread_num() << ", cuda initial is done." << " Start memcpy! AllTableElemets.size: " << AllTableElemets.size() << endl;


	int *ATE_myOPT = new int[AllTableElemets.size()];

	for (int i = 0; i < AllTableElemets.size(); i++){
		//gpuErrchk(hipMemcpyAsync(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
		gpuErrchk(hipMemcpy(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}


	//gpuErrchk(hipMemcpyAsync(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	
	gpuErrchk(hipMemsetAsync(dev_zeroVec, 0, powK*sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemsetAsync(dev_lock1, 0, AllTableElemets.size() * sizeof(int), streams[thread*batchSize]));
	//gpuErrchk(hipMemsetAsync(dev_lock2, 0, AllTableElemets.size() * sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemsetAsync(dev_ATE_NSsubsets_size, 0, AllTableElemets.size()*sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_counterVec, &counterVec[0], counterVec.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_blockDimSize, &blockDimSize[0], blockDimSize.size()*sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_divisor, &divisor[0], divisor.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_divisorComp, &divisorComp[0], powK * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
/*
	gpuErrchk(hipMemset(dev_zeroVec, 0, powK*sizeof(int)));
	gpuErrchk(hipMemcpy(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_counterVec, &counterVec[0], counterVec.size() * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_blockDimSize, &blockDimSize[0], blockDimSize.size()*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_divisor, &divisor[0], divisor.size() * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_divisorComp, &divisorComp[0], powK * sizeof(int), hipMemcpyHostToDevice));
*/	
	int b = 0;
	while (ii < blockCounterVec.size())
	{
		for (int tt = 0; tt < blockCounterVec[ii]; tt++)
		{
			//each stream process only one block at a time. Dispatch blocks within the same level to different streams.
			int subStream = tt % batchSize;
			//streamOffset is the global memory offset for one cpu thread, it returns the memory offset for the current stream.
			//int streamOffset = subStream * maxBlockLvlSize * maxInBlockLvlSize * powK;
			//configLvlOffset returns the memory offset for the block tt within the same stream.
			//Since each stream is allocated to the MAX memory space, only some memory are used. For example maxBlockLvlSize = 100, but we
			//have only 45 blocks in the current level. Thus, configLvlOffset helps find the exact memory offset.
			//int configLvlOffset = tt * maxInBlockLvlSize * powK;
			//This offset is used by arrays it, ss, and NS as temporary storage.
			//int totalOffset = streamOffset + configLvlOffset;

			if (allBlocks[b].mySUM == ii)
			{
#ifdef _DEVICE_DEBUG
				if (thread == 0){
					cout << "thread: " << thread << ", allBlocks[" << b << "]: ";
					for (int i=0; i<powK; i++)
					{
						cout << allBlocks[b].elm[i] << " ";
					}
					cout << "mySum: " << allBlocks[b].mySUM << endl;
					cout << "thread: " << thread << ", blockNoZero[" << b << "]: ";
					for (int i=0; i<allBlocksNoZero[b].elm.size(); i++)
					{
						cout << allBlocksNoZero[b].elm[i] << " ";
					}
					cout << endl;
				}
#endif
				//blockOffset returns the current block ID.
				int blockOffset = gpu_blockOffsetNoZero(&allBlocksNoZero[b].elm[0], &divisor[0],
												allBlocksNoZero[b].elm.size(), divisor.size());
				//configOffset returns the number of jobs before the current block. This is not memory offset.
				int configOffset = blockOffset * jobsPerBlock;

				//cout << "cpu: " << thread << ", blockOffset: " << blockOffset << ", configOffset: " << configOffset << endl;

				//allBlocks contains the block ID which is needed to calculate the beginning vector position in AllTableElemets.
/*				LaunchBlocks<<<1, 1, 0, streams[thread*batchSize+subStream]>>>(blockOffset,
						dev_ATE_elm, dev_counterVec, dev_roundVec, powK, thread,
						AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets,
						dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_myOPT, dev_ATE_myOptimalindex,
						dev_ATE_myMinNSVector, ii, maxSubsetsSize, dev_ATE_optVector, jobsPerBlock,
						dev_blockDimSize, configOffset, T, &it[totalOffset], &ss[totalOffset],
						&NS[totalOffset], dev_divisor, dev_divisorComp, divisor.size(), 
						counterVec.size(), tt, dev_lock1, optVectorSize);    */
				LaunchBlocks<<<1, 1, 0, streams[thread*batchSize+subStream]>>>(blockOffset,
						dev_ATE_elm, dev_counterVec, dev_roundVec, powK, thread,
						AllTableElemets.size(), dev_ATE_NSsubsets, dev_ATE_NSsubsets_size, 
						dev_zeroVec, dev_ATE_myOPT, dev_ATE_myOptimalindex, dev_ATE_myMinNSVector, 
						ii, maxSubsetsSize, jobsPerBlock, dev_blockDimSize, configOffset, T, 
						dev_divisor, dev_divisorComp, divisor.size(), counterVec.size(), tt, dev_lock1);
				//break;
				b++;
			}

		}

		//for (int j = 0; j < blockCounterVec[ii]; j++)
		for (int subStream = 0; subStream < batchSize; subStream++)
		{
			//int subStream = j % batchSize;
			hipStreamSynchronize(streams[thread*batchSize+subStream]);
		}

		indexomp += blockCounterVec[ii];
		ii++;
	}

/*********************  GPU code to update AllTableElement  ********************************/
	int *temp_NSsubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	//int *temp_Csubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	int *temp_myOPT = new int[AllTableElemets.size()];
	int *temp_myOptIndex = new int[AllTableElemets.size()];
	int *temp_myMinNSVector = new int[AllTableElemets.size() * powK];
	//int *temp_optVector = new int[AllTableElemets.size() * optVectorSize];

//	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << endl;

	gpuErrchk(hipMemcpyAsync(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	//gpuErrchk(hipMemcpyAsync(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myMinNSVector, dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	//gpuErrchk(hipMemcpyAsync(temp_optVector, dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
/*
	gpuErrchk(hipMemcpy(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myMinNSVector, dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_optVector, dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int), hipMemcpyDeviceToHost));
*/
	struct timeval t1, t2;
	gettimeofday(&t1, NULL);

	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp_myOPT[i];
		AllTableElemets[i].myOptimalindex = temp_myOptIndex[i];
		int begin = 0, end = maxSubsetsSize * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp_NSsubsets[(i * maxSubsetsSize) * powK], &temp_NSsubsets[(i * maxSubsetsSize + 1) * powK]));
			//AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp_Csubsets[(i * maxSubsetsSize) * powK], &temp_Csubsets[(i * maxSubsetsSize + 1) * powK]));
			begin += powK;
		}
		//AllTableElemets[i].optVector.insert(AllTableElemets[i].optVector.end(), &temp_optVector[i * optVectorSize], &temp_optVector[(i + 1) * optVectorSize]);
		AllTableElemets[i].myMinNSVector.insert(AllTableElemets[i].myMinNSVector.end(), &temp_myMinNSVector[i * powK], &temp_myMinNSVector[(i + 1) * powK]);
	}

	gettimeofday(&t2, NULL);
	//cout << "memory transfer to vectors Runtime: "  << t2.tv_sec - t1.tv_sec << endl;


	delete[] ATE_myOPT;
	delete[] temp_NSsubsets;
	//delete[] temp_Csubsets;
	delete[] temp_myOPT;
	delete[] temp_myOptIndex;
	delete[] temp_myMinNSVector;
	//delete[] temp_optVector;
	//hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	//hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	//hipFree(it);
	//hipFree(ss);
	//hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	//hipFree(dev_ATE_optVector_size);
	hipFree(dev_blockDimSize);
	hipFree(dev_divisor);
	hipFree(dev_lock1);
	//hipFree(dev_lock2);

	for (int i=0; i<batchSize; i++)
		hipStreamDestroy(streams[thread*batchSize+i]);
}


#else
void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, const int maxSumValue,
			vector<int> &counterVec, const int LongJobs_size, int *zeroVec, int *roundVec)
{
	hipSetDevice(0);

	const int thread = omp_get_thread_num();
	const int numThreads = omp_get_num_threads();
	const int batchSize = maxStreamNum/numThreads;

	for (int i=0; i < batchSize; i++)
		hipStreamCreate(&streams[thread*batchSize+i]);

	int *dev_ATE_elm = 0, *dev_ATE_myOPT = 0, *dev_ATE_myOptimalindex = 0, *dev_ATE_myMinNSVector = 0;
	int *dev_ATE_NSsubsets = 0, *dev_ATE_Csubsets = 0;
	int *dev_ATE_optVector = 0;
	int *dev_counterVec = 0;
	int *dev_ATE_NSsubsets_size = 0;
	int *dev_ATE_optVector_size = 0;
	int *dev_zeroVec = 0, *dev_roundVec = 0;
	int *it = 0, *ss = 0, *NS = 0;

    int ii=0;
    int indexomp=0;
    int maxSubsetsSize = 0;
	const int optVectorSize = 64;


//	InitGPUData(powK, LongJobs_size, AllTableElemets, zeroVec, roundVec, &counterVec[0], maxSubsetsSize, optVectorSize, counterVec.size(),
//				&dev_ATE_elm, &dev_ATE_myOPT, &dev_ATE_myOptimalindex, &dev_ATE_myMinNSVector, &dev_ATE_NSsubsets, &dev_ATE_Csubsets,
//				&dev_ATE_optVector, &dev_counterVec, &dev_ATE_NSsubsets_size, &dev_ATE_optVector_size, &dev_zeroVec, &dev_roundVec, &it, &ss, &NS);

	int maxIndex = AllTableElemets.size() - 1;
	int maxCounterVec = 0;
	vector<int> temp;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}

	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
//		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}

	maxSubsetsSize = 128;


	for (int i = 0; i < counterVec.size(); i++)
	{
		if (counterVec[i] > maxCounterVec)
			maxCounterVec = counterVec[i];
	}

//	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;

	//arrays on device
	gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, counterVec.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&it, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&ss, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&NS, batchSize * maxCounterVec * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));


    //*********** For gpu_sameVector and gpu_genSubConfigs ****************************************************
    int *ifsame;
    int *dev_subConfigSize;
    int *dev_countC, *dev_countNS;
    gpuErrchk(hipMalloc((void**)&ifsame, batchSize * maxCounterVec * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_subConfigSize, batchSize * maxCounterVec * powK * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_countC, AllTableElemets.size() * maxSubsetsSize * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_countNS, AllTableElemets.size() * maxSubsetsSize * sizeof(int)));
	gpuErrchk(hipMemsetAsync(dev_countC, 0, AllTableElemets.size() * maxSubsetsSize *sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemsetAsync(dev_countNS, 0, AllTableElemets.size() * maxSubsetsSize *sizeof(int), streams[thread*batchSize]));
	//*********************************************************************************************************


	//cout << "thread: " << omp_get_thread_num() << ", cuda initial is done." << " Start memcpy! AllTableElemets.size: " << AllTableElemets.size() << endl;


	int *ATE_myOPT = new int[AllTableElemets.size()];

	for (int i = 0; i < AllTableElemets.size(); i++){
		gpuErrchk(hipMemcpyAsync(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}

	//gpuErrchk(hipMemcpyAsync(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemsetAsync(dev_zeroVec, 0, powK*sizeof(int), streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_counterVec, &counterVec[0], counterVec.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice, streams[thread*batchSize]));


//	cout << ", LongJob size: " << LongJobs_size << ", maxSumValue: " << maxSumValue << endl;

	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 256;
		int bSize = 1;

		for (int ptr=0; ptr < batchSize; ptr++)
		{
			if (ii < maxSumValue+1)
			{
				if (tSize < counterVec[ii]){
					bSize = (tSize + counterVec[ii] - 1) / tSize;
				}

				int sizeOffset = ptr * maxCounterVec * powK;
//		std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", indexomp: " << indexomp << std::endl;
//				gpu_generate2<<<bSize, tSize, 0, streams[thread*batchSize+ptr]>>>(maxSubsetsSize,
//						 dev_ATE_elm, powK, dev_ATE_Csubsets, dev_ATE_NSsubsets, dev_roundVec, T,
//						 dev_ATE_NSsubsets_size, dev_counterVec, ii, indexomp,
//						 dev_subConfigSize, dev_countC, dev_countNS);

				gpu_generate2<<<bSize, tSize, 0, streams[thread*batchSize+ptr]>>>(maxSubsetsSize,
							dev_ATE_elm, powK, dev_ATE_Csubsets, dev_ATE_NSsubsets, dev_roundVec,
							T, it, ss, NS, dev_ATE_NSsubsets_size, dev_counterVec, ii, indexomp);

				FindOPT<<<bSize, tSize, 0, streams[thread*batchSize+ptr]>>>(dev_ATE_elm,
													dev_counterVec, indexomp, dev_roundVec, powK,
													AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets,
													dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_optVector,
													dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex,
													dev_ATE_myMinNSVector, ii, maxSubsetsSize, optVectorSize,
													dev_countC, dev_countNS);

//				FindOPT<<<bSize, tSize, 0, streams[thread*batchSize+ptr]>>>(dev_ATE_elm,
//									dev_counterVec, indexomp, dev_roundVec, T, k, powK,
//									AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets,
//									dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_optVector,
//									dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex,
//									dev_ATE_myMinNSVector, ii, &it[sizeOffset], &ss[sizeOffset],
//									&NS[sizeOffset], maxSubsetsSize, optVectorSize);

				hipStreamSynchronize(streams[thread*batchSize+ptr]);

				indexomp+=counterVec[ii];
				ii++;
			}
		}
    }

/*********************  GPU code to update AllTableElement  ********************************/
	int *temp_NSsubsets, *temp_Csubsets, *temp_myOPT, *temp_myOptIndex, *temp_myMinNSVector, *temp_optVector;
	//int *temp_countC, *temp_countNS;
	temp_NSsubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_Csubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_myOPT = new int[AllTableElemets.size()];
	temp_myOptIndex = new int[AllTableElemets.size()];
	temp_myMinNSVector = new int[AllTableElemets.size() * powK];
	temp_optVector = new int[AllTableElemets.size() * optVectorSize];
	//temp_countC = new int[AllTableElemets.size() * maxSubsetsSize];
	//temp_countNS = new int[AllTableElemets.size() * maxSubsetsSize];

//	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << endl;

	gpuErrchk(hipMemcpyAsync(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_myMinNSVector, dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	gpuErrchk(hipMemcpyAsync(temp_optVector, dev_ATE_optVector, AllTableElemets.size() * optVectorSize * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	//gpuErrchk(hipMemcpyAsync(temp_countC, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));
	//gpuErrchk(hipMemcpyAsync(temp_countNS, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * sizeof(int), hipMemcpyDeviceToHost, streams[thread*batchSize]));

	struct timeval t1, t2;
	gettimeofday(&t1, NULL);

	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp_myOPT[i];
		AllTableElemets[i].myOptimalindex = temp_myOptIndex[i];
		int begin = 0, end = maxSubsetsSize * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp_NSsubsets[(i * maxSubsetsSize) * powK], &temp_NSsubsets[(i * maxSubsetsSize + 1) * powK]));
			AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp_Csubsets[(i * maxSubsetsSize) * powK], &temp_Csubsets[(i * maxSubsetsSize + 1) * powK]));
			begin += powK;
		}
		AllTableElemets[i].optVector.insert(AllTableElemets[i].optVector.end(), &temp_optVector[i * optVectorSize], &temp_optVector[(i + 1) * optVectorSize]);
		AllTableElemets[i].myMinNSVector.insert(AllTableElemets[i].myMinNSVector.end(), &temp_myMinNSVector[i * powK], &temp_myMinNSVector[(i + 1) * powK]);
	}

	gettimeofday(&t2, NULL);
	cout << "memory transfer to vectors Runtime: "  << t2.tv_sec - t1.tv_sec << endl;


	delete[] ATE_myOPT;
	delete[] temp_NSsubsets;
	delete[] temp_Csubsets;
	delete[] temp_myOPT;
	delete[] temp_myOptIndex;
	delete[] temp_myMinNSVector;
	delete[] temp_optVector;
	//delete[] temp_countC;
	//delete[] temp_countNS;
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	hipFree(dev_ATE_optVector_size);

	hipFree(ifsame);
	hipFree(dev_subConfigSize);
	hipFree(dev_countC);
	hipFree(dev_countNS);

	for (int i=0; i<batchSize; i++)
		hipStreamDestroy(streams[thread*batchSize+i]);
}
#endif
