#include "hip/hip_runtime.h"
__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(const int *Ntemp, int *it, int Ntemp_size){
#pragma unroll
	for (int i = 0, size = Ntemp_size; i != size; ++i) {
		const int index = size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B)
{
	int summ=0.0;
#pragma unroll
	for(int i=0; i<(Pow(k,2)); i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, const int T, const int powk){
	
	//Ntemp_size = pow(k,2)
	
	//vector<int> it(Ntemp.size(), 0);
	int it[Ntemp_size];
	int counter = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
		it[i] = 0;
    do {
        int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s,roundVec);
        if(sSum <= T)
        {
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counter*Ntemp_size + i] = s[i];
			}
            //Ctemp.push_back(s);
            
            int NS[Ntemp_size];
#pragma unroll
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size))
                continue;
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counter * Ntemp_size + i] = NS[i];
			}
            //NMinusStemp.push_back(NS);
        }
        counter++;
    }while (gpu_increase(Ntemp, it, Ntemp_size));
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, const int k, const int powK, const int dev_AllTableElemets_size,
						int *dev_ATE_Csubsets, int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int Cwhole_size, int *dev_zeroVec, int *dev_ATE_optVector, int *dev_ATE_optVector_size,
						int *dev_ATE_myOPT, int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector){		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
		int j = thread + indexomp;
		if (j < dev_counterVec[i] + indexomp){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
            
            gpu_generate2(&dev_ATE_elm[j * powK], powK, dev_ATE_Csubsets, dev_ATE_NSsubsets, dev_roundVec, T, powK);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * powK + optVecIndex] = 0;
                    optVecIndex++;
                    break;
                }
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], dev_ATE_elm[j * powK], powK) )
                    break;
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < dev_AllTableElemets_size; r++)
                {

                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 
                    if (dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], dev_ATE_elm[r * powK], powK)
                    {
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * powK + optVecIndex] = dev_ATE_myOPT[r];
                        optVecIndex++;
                        break;
                    }
                }
            }
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
#pragma unroll
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
       //         cout << AllTableElemets[j].optVector[pp]<<" ";
//                if(AllTableElemets[j].optVector[pp] < minn)
				if (dev_ATE_optVector[j * powK + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * powK + pp];
                    myOptimalindex=pp;
                }
            }
          //  cout << endl;
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
				// dev_ATE_NSsubsets_size[j], does this vector store all same value for sizes?
				hipMemcpy(&dev_ATE_myMinNSVector[j * vectorSize], &dev_ATE_NSsubsets[(j * Cwhole_size + myOptimalindex) * powK], powK, hipMemcpyDeviceToDevice);
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }
		}//end if (j)
	}//end FindOPT()
