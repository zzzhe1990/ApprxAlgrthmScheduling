#include "hip/hip_runtime.h"
#include "DPCUDA.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

int *dev_ATE_elm, *dev_ATE_myOPT, *dev_ATE_myOptimalindex, *dev_ATE_myMinNSVector;
	int *dev_ATE_NSsubsets, *dev_ATE_Csubsets;
	int *dev_ATE_optVector;
	int *dev_counterVec;
	int *dev_ATE_NSsubsets_size;
	int *dev_ATE_optVector_size;
	int *dev_zeroVec, *dev_roundVec;
	int *it, *ss, *NS;
	
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
	
void InitGPUData(int AllTableElemets_size, int Cwhole_size, int powK, int LongJobs_size, 
				 vector<DynamicTable> &AllTableElemets, int *zeroVec, int *roundVec, int *counterVec)
{
	//arrays on device
	gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets_size * Cwhole_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets_size * Cwhole_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets_size * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets_size * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, (LongJobs_size + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&it, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&ss, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&NS, AllTableElemets_size * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets_size * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets_size * sizeof(int)));
    
	int *ATE_NSsubsets_size = new int[AllTableElemets_size];
	int *ATE_optVector_size = new int[AllTableElemets_size];
	for (int i = 0; i < AllTableElemets_size; i++){
//		ATE_NSsubsets_size[i] = AllTableElemets[i].NSsubsets.size();
//		ATE_optVector_size[i] = AllTableElemets[i].optVector.size();
		gpuErrchk(hipMemcpy(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice));
	}
    
	gpuErrchk(hipMemcpy(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_NSsubsets_size, ATE_NSsubsets_size, AllTableElemets_size * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_optVector_size, ATE_optVector_size, AllTableElemets_size * sizeof(int), hipMemcpyHostToDevice));
	
	delete(ATE_NSsubsets_size);
	delete(ATE_optVector_size);
}


__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(const int *Ntemp, int *it, int Ntemp_size){
#pragma unroll
	for (int i = 0, size = Ntemp_size; i != size; ++i) {
		const int index = size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, const int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, 
							  const int T, const int powK, int *it, int *s, int *NS, int *NSsubsets_size){
	
	//Ntemp_size = pow(k,2)
	
	//vector<int> it(Ntemp.size(), 0);
	//int it[Ntemp_size];
	int counter = 0;
	int NS_counter = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
		it[i] = 0;
    do {
        //int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s, dev_roundVec, powK);
        if(sSum <= T)
        {
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counter*Ntemp_size + i] = s[i];
			}
            //Ctemp.push_back(s);
            
            //int NS[Ntemp_size];
#pragma unroll
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size)){
				for (int i = 0; i < Ntemp_size; i++)
				{
					NMinusStemp[counter * Ntemp_size + i] = -1;
				}
                continue;
			}
#pragma unroll
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counter * Ntemp_size + i] = NS[i];
				NS_counter++;
			}
            //NMinusStemp.push_back(NS);
        }
        counter++;
    }while (gpu_increase(Ntemp, it, Ntemp_size));
    
    NSsubsets_size[0] = NS_counter;
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, 
						const int k, const int powK, const int dev_AllTableElemets_size, int *dev_ATE_Csubsets, 
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int Cwhole_size, int *dev_zeroVec, 
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it, 
						int *s, int *NS){		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
//			printf("thread: %d, i: %d, dev_counterVec[i]: %d\n", thread, i, dev_counterVec[i]);
		int j = thread + indexomp;
		if (thread < dev_counterVec[i]){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
            
            gpu_generate2(&dev_ATE_elm[j * powK], powK, &dev_ATE_Csubsets[j * Cwhole_size * powK], &dev_ATE_NSsubsets[j * Cwhole_size * powK],
						  dev_roundVec, T, powK, &it[j * powK], &s[j * powK], &NS[j * powK], &dev_ATE_NSsubsets_size[j]);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
//			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
			for(int h=0; h < Cwhole_size; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * powK + optVecIndex] = 0;
                    optVecIndex++;
                    dev_ATE_optVector_size[j] = optVecIndex;
                    break;
                }
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], &dev_ATE_elm[j * powK], powK) ){
                    dev_ATE_optVector_size[j] = optVecIndex;
                    break;
				}
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < dev_AllTableElemets_size; r++)
                {

                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 
                    if (gpu_sameVectors(&dev_ATE_NSsubsets[(j * Cwhole_size + h) * powK], &dev_ATE_elm[r * powK], powK))
                    {
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * powK + optVecIndex] = dev_ATE_myOPT[r];
                        optVecIndex++;
						dev_ATE_optVector_size[j] = optVecIndex;
                        break;
                    }
                }
            }
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
#pragma unroll
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
       //         cout << AllTableElemets[j].optVector[pp]<<" ";
//                if(AllTableElemets[j].optVector[pp] < minn)
				if (dev_ATE_optVector[j * powK + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * powK + pp];
                    myOptimalindex=pp;
                }
            }
          //  cout << endl;
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
				for (int i = 0; i < powK; i++){
					dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * Cwhole_size + myOptimalindex) * powK + i];
				}
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }
            
		}//end if (j)
	}//end FindOPT()
/*
void gpu_DP(vector<DynamicTable> &AllTableElemets, int *dev_ATE_elm, int *dev_counterVec, int *dev_roundVec, 
			const int T, const int k, const int powK, const int dev_AllTableElemets_size,
			int *dev_ATE_Csubsets, int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, 
			int Cwhole_size, int *dev_zeroVec, int *dev_ATE_optVector, int *dev_ATE_optVector_size,
			int *dev_ATE_myOPT, int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, 
			int *it, int *s, int *NS, const int maxSumValue, vector<int> &counterVec)*/
void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, 
			const int dev_AllTableElemets_size, int Cwhole_size, const int maxSumValue, 
			vector<int> &counterVec, const int LongJobs_size)
{
    int ii=0;
    int indexomp=0;

//	for (int i = 0; i < maxSumValue+1; i++){
//		std::cout << "counterVec[" << i << "]: " << counterVec[i] << std::endl;
//	}

	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 32;
		int bSize = 1;
		if (tSize < counterVec[ii]){
			bSize = (tSize + counterVec[ii] - 1) / tSize;
		}
		//std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", bSize: " << bSize << ", tSize: " << tSize << std::endl;
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec, T, k, powK, 
								  dev_AllTableElemets_size, dev_ATE_Csubsets, dev_ATE_NSsubsets, 
								  dev_ATE_NSsubsets_size, Cwhole_size, dev_zeroVec, dev_ATE_optVector, 
								  dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex, 
								  dev_ATE_myMinNSVector, ii, it, ss, NS);
        
               
        gpuErrchk(hipMemcpy(&counterVec[0], dev_counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyDeviceToHost));
        indexomp+=counterVec[ii];
        ii++;
    }
    
//GPU code to update AllTableElement
	int *temp1, *temp2, *temp3, *temp4;
	temp1 = new int[AllTableElemets.size() * Cwhole_size * powK];
	temp2 = new int[AllTableElemets.size() * Cwhole_size * powK];
	temp3 = new int[AllTableElemets.size()];
	temp4 = new int[AllTableElemets.size()];
	/*
	for(int i=0; i < AllTableElemets.size(); i++){
		AllTableElemets[i].NSsubsets.resize(Cwhole_size);
		AllTableElemets[i].Csubsets.resize(Cwhole_size);
		AllTableElemets[i].optVector.resize(powK);
		for (int j = 0; j < Cwhole_size; j++){
			AllTableElemets[i].NSsubsets[j].resize(powK);
			AllTableElemets[i].Csubsets[j].resize(powK);
//			hipMemcpy(&AllTableElemets[i].NSsubsets[j][0], &dev_ATE_NSsubsets[(i * Cwhole_size + j) * powK], powK, hipMemcpyDeviceToHost);
//			hipMemcpy(&AllTableElemets[i].Csubsets[j][0], &dev_ATE_Csubsets[(i * Cwhole_size + j) * powK], powK, hipMemcpyDeviceToHost);
			gpuErrchk(hipMemcpy(temp1, &dev_ATE_NSsubsets[(i * Cwhole_size + j) * powK], powK * sizeof(int), hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(temp2, &dev_ATE_Csubsets[(i * Cwhole_size + j) * powK], powK * sizeof(int), hipMemcpyDeviceToHost));
//			std::copy(temp1, temp1+powK, std::back_inserter(AllTableElemets[i].NSsubsets));
//			std::copy(temp2, temp2+powK, std::back_inserter(AllTableElemets[i].Csubsets));
//			AllTableElemets[i].NSsubsets[j].insert(AllTableElemets[i].NSsubsets[j].begin(), temp1, temp1+powK);
//			AllTableElemets[i].Csubsets[j].insert(AllTableElemets[i].Csubsets[j].begin(), temp2, temp2+powK);
			for (int p = 0; p < powK; p++){
				AllTableElemets[i].NSsubsets[j][p] = temp1[p];
				AllTableElemets[i].Csubsets[j][p] = temp2[p];
			}
			std::cout << "memcpy of NSsubsets and Csubsets for dimension: " << j << " of AllTableElemets[" << i <<"] is done." << std::endl;
		}
		gpuErrchk(hipMemcpy(&AllTableElemets[i].optVector[0], &dev_ATE_optVector[i * powK], powK * sizeof(int), hipMemcpyDeviceToHost));
		//Csubsets[Cwhole.size()][powK]
	}
	*/
	gpuErrchk(hipMemcpy(temp1, dev_ATE_NSsubsets, AllTableElemets.size() * Cwhole_size * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp2, dev_ATE_Csubsets, AllTableElemets.size() * Cwhole_size * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp3, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp4, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	
	std::cout << "temp1 temp2 is done, AllTableElemets.size: " << AllTableElemets.size() << std::endl;
	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp3[i];
		AllTableElemets[i].myOptimalindex = temp4[i];
		AllTableElemets[i].NSsubsets.resize(Cwhole_size);
		AllTableElemets[i].Csubsets.resize(Cwhole_size);
		AllTableElemets[i].optVector.resize(powK);
		int begin = 0, end = Cwhole_size * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp1[(i * Cwhole_size) * powK], &temp1[(i * Cwhole_size + 1) * powK]));
			AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp2[(i * Cwhole_size) * powK], &temp2[(i * Cwhole_size + 1) * powK]));
			begin += powK;
		}
		gpuErrchk(hipMemcpy(&AllTableElemets[i].optVector[0], &dev_ATE_optVector[i * powK], powK * sizeof(int), hipMemcpyDeviceToHost));
	}
	
	
	free(temp1);
	free(temp2);
	free(temp3);
	free(temp4);
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	hipFree(dev_ATE_optVector_size);
}
