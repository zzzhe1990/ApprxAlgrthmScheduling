#include "hip/hip_runtime.h"
#include "DPCUDA.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

int *dev_ATE_elm, *dev_ATE_myOPT, *dev_ATE_myOptimalindex, *dev_ATE_myMinNSVector;
	int *dev_ATE_NSsubsets, *dev_ATE_Csubsets;
	int *dev_ATE_optVector;
	int *dev_counterVec;
	int *dev_ATE_NSsubsets_size;
	int *dev_ATE_optVector_size;
	int *dev_zeroVec, *dev_roundVec;
	int *it, *ss, *NS;
	
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
	
void InitGPUData(int powK, int LongJobs_size, vector<DynamicTable> &AllTableElemets, 
				 int *zeroVec, int *roundVec, int *counterVec, int &maxSubsetsSize)
{
	cout << "At the beginning of InitGPUData. maxSubsetsSize: " << maxSubsetsSize << endl;
	
	int maxIndex = AllTableElemets.size() - 1;
	vector<int> temp;
	cout << "check valid element AllTableElemets[" << maxIndex << "].elm[powK-1]: " << AllTableElemets[maxIndex].elm[powK-1] << endl;
	cout << "This is the current AllTableElemets[" << maxIndex << "].elm: " << endl;
//	for (vector<int>::const_iterator pt = AllTableElemets[maxIndex].elm.end(); pt != AllTableElemets[maxIndex].elm.begin(); --pt)
	for (int p = powK-1; p >= 0; p--)
	{
		cout << AllTableElemets[maxIndex].elm[p] << " ";
		if (AllTableElemets[maxIndex].elm[p] != 0)
			temp.push_back(AllTableElemets[maxIndex].elm[p]);
	}
	cout << endl;
	
	for (int i = 0; i < temp.size(); i++)
	{
		int a = 1;
		for (int j = 0; j < i + 1; j++)
		{
			a *= temp[j];
		}
		cout << "Update maxSubsetsSize, current: " << maxSubsetsSize << ", a: " << a <<endl;
		maxSubsetsSize += a;
	}
	
	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << ", tempSize: " << temp.size() << endl;
	
	//arrays on device
	gpuErrchk(hipMalloc((void**)&dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_elm, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myMinNSVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOPT, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_optVector, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_counterVec, (LongJobs_size + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_zeroVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_roundVec, (powK) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&it, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&ss, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&NS, AllTableElemets.size() * powK * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_ATE_NSsubsets_size, AllTableElemets.size() * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_ATE_optVector_size, AllTableElemets.size() * sizeof(int)));
    
	int *ATE_optVector_size = new int[AllTableElemets.size()];
	int *ATE_myOPT = new int[AllTableElemets.size()];
	for (int i = 0; i < AllTableElemets.size(); i++){
//		ATE_optVector_size[i] = AllTableElemets[i].optVector.size();
		gpuErrchk(hipMemcpy(&dev_ATE_elm[i * powK], &AllTableElemets[i].elm[0], powK * sizeof(int), hipMemcpyHostToDevice));
		ATE_myOPT[i] = AllTableElemets[i].myOPT;
	}
	
	gpuErrchk(hipMemcpy(dev_zeroVec, zeroVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_roundVec, roundVec, powK*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_counterVec, counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_optVector_size, ATE_optVector_size, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_ATE_myOPT, ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyHostToDevice));
	
	delete(ATE_optVector_size);
	delete(ATE_myOPT);
}


__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}	
	}
	return same;
}

__device__ int gpu_increase(const int *Ntemp, int *it, int Ntemp_size){
#pragma unroll
	for (int i = 0, size = Ntemp_size; i != size; ++i) {
		const int index = size - 1 - i;
		++it[index];
		if (it[index] > Ntemp[index]) {
			it[index] = 0;
		} else {
			return 1;
		}
	}
	return 0;
}

__device__ int gpu_sumFun(int *A, int *B, const int powK)
{
	int summ=0.0;
#pragma unroll
	for(int i=0; i<powK; i++)
	{
		summ= summ + A[i]*B[i];
	}
	return summ;
}

__device__ void gpu_generate2(int *Ntemp, const int Ntemp_size, int *Ctemp, int *NMinusStemp, int *dev_roundVec, 
							  const int T, const int powK, int *it, int *s, int *NS, int *subsets_size){
	
	//Ntemp_size = pow(k,2)
	
	//vector<int> it(Ntemp.size(), 0);
	//int it[Ntemp_size];
	int counter = 0;
#pragma unroll
	for (int i = 0; i < Ntemp_size; i++)
		it[i] = 0;
    do {
        //int s[Ntemp_size];
#pragma unroll
        for (int i = 0; i < Ntemp_size; i++)
        {
			s[i] = it[i];
		}
        //for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
        //{
        //    s.push_back(*i);
        //}
        //Cwhole.push_back(s);
        int sSum=gpu_sumFun(s, dev_roundVec, powK);
        if(sSum <= T)
        {
			for (int i = 0; i < Ntemp_size; i++)
			{
				Ctemp[counter*Ntemp_size + i] = s[i];
			}
            //Ctemp.push_back(s);
            
            //int NS[Ntemp_size];
            for(int j=0; j<powK; j++)
            {
                NS[j] = Ntemp[j]-s[j];
            }
            if(gpu_sameVectors(NS, Ntemp, Ntemp_size)){
                continue;
			}
			
			for (int i = 0; i < Ntemp_size; i++)
			{
				NMinusStemp[counter * Ntemp_size + i] = NS[i];
			}
            //NMinusStemp.push_back(NS);
			counter++;
        }
    }while (gpu_increase(Ntemp, it, Ntemp_size));
    
    *subsets_size = counter;
}

//for(int j=indexomp;j< (counterVec[i] + indexomp) ;j++)			// this is to determine the job for each level
		
__global__ void FindOPT(int *dev_ATE_elm, int *dev_counterVec, int indexomp, int *dev_roundVec, const int T, 
						const int k, const int powK, const int AllTableElemets_size, int *dev_ATE_Csubsets, 
						int *dev_ATE_NSsubsets, int *dev_ATE_NSsubsets_size, int *dev_zeroVec, 
						int *dev_ATE_optVector, int *dev_ATE_optVector_size, int *dev_ATE_myOPT, 
						int *dev_ATE_myOptimalindex, int *dev_ATE_myMinNSVector, const int i, int *it, 
						int *s, int *NS, const int maxSubsetsSize){		
		int thread = blockDim.x * blockIdx.x + threadIdx.x;
//			printf("thread: %d, i: %d, dev_counterVec[i]: %d\n", thread, i, dev_counterVec[i]);
		int j = thread + indexomp;
		if (thread < dev_counterVec[i]){
            //vector<vector<int> > Ctemp;
            //vector<vector<int> > NMinusStemp;
            //vector<vector<int> > Cwhole;
            //generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
            
            gpu_generate2(&dev_ATE_elm[j * powK], powK, &dev_ATE_Csubsets[j * maxSubsetsSize * powK], &dev_ATE_NSsubsets[j * maxSubsetsSize * powK],
						  dev_roundVec, T, powK, &it[j * powK], &s[j * powK], &NS[j * powK], &dev_ATE_NSsubsets_size[j]);     //dev_ATE_elm_size[j] = Ntemp.size() = powK
			
			__syncthreads();
//            AllTableElemets[j].NSsubsets=NMinusStemp; 	//ni-si
//            AllTableElemets[j].Csubsets=Ctemp;		//configurations
            
//            for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j], 
																	// NSTableElements is the table for all previous OPT. Find all subsets(dependency) of selected job
			int optVecIndex = 0;
//			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
			int hit1 = 0, hit2 = 0, hit3 = 0;
			for(int h=0; h < dev_ATE_NSsubsets_size[j]; h++)
            {
//                if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], dev_zeroVec, powK))
                {
                    //AllTableElemets[j].optVector.push_back(0);
                    dev_ATE_optVector[j * powK + optVecIndex] = 0;
                    optVecIndex++;
                    dev_ATE_optVector_size[j] = optVecIndex;
                    hit1++;
                    break;
                }
                //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) 
																			//( the one that we are doing operation for it ) ----> break (not interested )
																			// check if it is itself, if yes, ignore OPT of job itself.
				if(gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[j * powK], powK) ){
                    dev_ATE_optVector_size[j] = optVecIndex;
                    hit2++;
                    break;
				}
                //for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
																//dependencies may not be consectively stored in the table, so have to go through the whole
																//table (AllTableElemets) and find them (matched to AllTableElemets[j].NSsubsets[h]).
				for (int r = 0; r < AllTableElemets_size; r++)
                {
//					if (j == 0)
//						printf("thread: %d, j: %d, r: %d, myOPT[%d]: %d\n", thread, j, r, r, dev_ATE_myOPT[r]);
                    //if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break 
                    if (gpu_sameVectors(&dev_ATE_NSsubsets[(j * maxSubsetsSize + h) * powK], &dev_ATE_elm[r * powK], powK))
                    {
					//	if (thread%100 == 0){
					//		printf("thread: %d, j: %d, NSsubsets seg:", thread, j);
					//		for (int i = 0; i < powK; i++)
					//			printf(" %d", dev_ATE_NSsubsets[(j*maxSubsetsSize+h)*powK+i]);
					//		printf("\n");
					//		printf("thread: %d, j: %d, Csubsets seg:", thread, j);
					//		for (int i = 0; i < powK; i++)
					//			printf(" %d", dev_ATE_elm[r*powK+i]);
					//		printf("\n");
					//	}
                        //AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
                        dev_ATE_optVector[j * powK + optVecIndex] = dev_ATE_myOPT[r];
                        optVecIndex++;
						dev_ATE_optVector_size[j] = optVecIndex;
						hit3++;
                        break;
                    }
                }
            }
            int minn = 100000;
            int myOptimalindex;
            //for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)			// find out the OPT from all dependencies.
            for (int pp = 0; pp < dev_ATE_optVector_size[j]; pp++)
            {
       //         cout << AllTableElemets[j].optVector[pp]<<" ";
//                if(AllTableElemets[j].optVector[pp] < minn)
				//if (thread == 0)
				//	printf("j: %d, thread: %d, AllTableElemets[%d].optVector[%d]: %d\n", j, thread, j, pp, dev_ATE_optVector[j*powK+pp]);
				if (dev_ATE_optVector[j * powK + pp] < minn)
                {
//                    minn=AllTableElemets[j].optVector[pp];
					minn = dev_ATE_optVector[j * powK + pp];
                    myOptimalindex=pp;
                }
            }
          //  cout << endl;
            int optTemp=minn+1;
//            AllTableElemets[j].myOPT=optTemp;
			dev_ATE_myOPT[j] = optTemp;
//            AllTableElemets[j].myOptimalindex=myOptimalindex;
			dev_ATE_myOptimalindex[j] = myOptimalindex;
            
//            if(AllTableElemets[j].NSsubsets.size()>0)
			if (dev_ATE_NSsubsets_size[j] > 0)
            {
//                AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
#pragma unroll
				for (int i = 0; i < powK; i++){
					dev_ATE_myMinNSVector[j * powK + i] = dev_ATE_NSsubsets[(j * maxSubsetsSize + myOptimalindex) * powK + i];
				}
				//dev_ATE_myMinNSVector[j] = dev_ATE_NSsubsets[(j * CWhole.SIZE + myOptimalindex) * pow(k,2)];
            }
            
		}//end if (j)
	}//end FindOPT()

void gpu_DP(vector<DynamicTable> &AllTableElemets, const int T, const int k, const int powK, const int maxSumValue, 
			vector<int> &counterVec, const int LongJobs_size, int *zeroVec, int *roundVec)
{
    int ii=0;
    int indexomp=0;
    int maxSubsetsSize = 0;
	
	InitGPUData(powK, LongJobs_size, AllTableElemets, zeroVec, roundVec, &counterVec[0], maxSubsetsSize);
	
//	for (int i = 0; i < maxSumValue+1; i++){
//		std::cout << "counterVec[" << i << "]: " << counterVec[i] << std::endl;
//	}
	cout << ", LongJob size: " << LongJobs_size << ", maxSumValue: " << maxSumValue << endl;
	while (ii < maxSumValue+1)		//number of levels = number of jobs + 1
    {
		int tSize = 32;
		int bSize = 1;
		if (tSize < counterVec[ii]){
			bSize = (tSize + counterVec[ii] - 1) / tSize;
		}
		//std::cout << "counterVec[" << ii << "]: " << counterVec[ii] << ", bSize: " << bSize << ", tSize: " << tSize << std::endl;
		FindOPT<<<bSize, tSize>>>(dev_ATE_elm, dev_counterVec, indexomp, dev_roundVec, T, k, powK, 
								  AllTableElemets.size(), dev_ATE_Csubsets, dev_ATE_NSsubsets, 
								  dev_ATE_NSsubsets_size, dev_zeroVec, dev_ATE_optVector, 
								  dev_ATE_optVector_size, dev_ATE_myOPT, dev_ATE_myOptimalindex, 
								  dev_ATE_myMinNSVector, ii, it, ss, NS, maxSubsetsSize);
           
//        gpuErrchk(hipMemcpy(&counterVec[0], dev_counterVec, (LongJobs_size + 1) * sizeof(int), hipMemcpyDeviceToHost));
        indexomp+=counterVec[ii];
        ii++;
    }
    hipDeviceSynchronize();
//GPU code to update AllTableElement
	int *temp_NSsubsets, *temp_Csubsets, *temp_myOPT, *temp_myOptIndex;
	temp_NSsubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_Csubsets = new int[AllTableElemets.size() * maxSubsetsSize * powK];
	temp_myOPT = new int[AllTableElemets.size()];
	temp_myOptIndex = new int[AllTableElemets.size()];
	
	cout << "FindOPT recursion is done. Start memcpy from Device to Host." << endl;
	cout << "AllTableSize: " << AllTableElemets.size() << ", maxSubsetsSize: " << maxSubsetsSize << ", powK: " << powK << endl;
	
	gpuErrchk(hipMemcpy(temp_NSsubsets, dev_ATE_NSsubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_Csubsets, dev_ATE_Csubsets, AllTableElemets.size() * maxSubsetsSize * powK * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOPT, dev_ATE_myOPT, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp_myOptIndex, dev_ATE_myOptimalindex, AllTableElemets.size() * sizeof(int), hipMemcpyDeviceToHost));
	
	std::cout << "memcpy from device to host are done, AllTableElemets.size: " << AllTableElemets.size() << std::endl;
/*	
	cout << "myOPT: ";
	for (int i = 0; i < AllTableElemets.size(); i++){
		cout << temp3[i] << ", ";
	}
	cout << endl;
	
	cout << "NSsubsets of AllTableElemets[0]: ";
	for (int i = 0; i < maxSubsetsSize * powK; i++){
		cout << temp1[i] << ", ";
	}
	cout << endl << "Csubsets of AllTableElemets[0]: ";
	for (int i = 0; i < maxSubsetsSize * powK; i++) {
		cout << temp2[i] << ", ";
	}
	cout << endl;
*/	
	for (int i = 0; i < AllTableElemets.size(); i++)
	{
		AllTableElemets[i].myOPT = temp_myOPT[i];
		AllTableElemets[i].myOptimalindex = temp_myOptIndex[i];
		AllTableElemets[i].NSsubsets.resize(maxSubsetsSize);
		AllTableElemets[i].Csubsets.resize(maxSubsetsSize);
		AllTableElemets[i].optVector.resize(powK);
		int begin = 0, end = maxSubsetsSize * powK;
		while (begin != end)
		{
			AllTableElemets[i].NSsubsets.push_back(std::vector<int>(&temp_NSsubsets[(i * maxSubsetsSize) * powK], &temp_NSsubsets[(i * maxSubsetsSize + 1) * powK]));
			AllTableElemets[i].Csubsets.push_back(std::vector<int>(&temp_Csubsets[(i * maxSubsetsSize) * powK], &temp_Csubsets[(i * maxSubsetsSize + 1) * powK]));
			begin += powK;
		}
		gpuErrchk(hipMemcpy(&AllTableElemets[i].optVector[0], &dev_ATE_optVector[i * powK], powK * sizeof(int), hipMemcpyDeviceToHost));
	}
	
	
	free(temp_NSsubsets);
	free(temp_Csubsets);
	free(temp_myOPT);
	free(temp_myOptIndex);
	hipFree(dev_ATE_Csubsets);
	hipFree(dev_ATE_NSsubsets);
	hipFree(dev_ATE_elm);
	hipFree(dev_ATE_myMinNSVector);
	hipFree(dev_ATE_myOPT);
	hipFree(dev_ATE_myOptimalindex);
	hipFree(dev_ATE_optVector);
	hipFree(dev_counterVec);
	hipFree(dev_zeroVec);
	hipFree(dev_roundVec);
	hipFree(it);
	hipFree(ss);
	hipFree(NS);
	hipFree(dev_ATE_NSsubsets_size);
	hipFree(dev_ATE_optVector_size);
}
