#include "hip/hip_runtime.h"
#include "DPCuda.hh"

GpuDynamicTable *h_AllGpuTableElements;
GpuDynamicTable *d_AllGpuTableElements;
int             *h_counterVec;
int							*d_counterVec;

__global__
void gpu_dpFunction(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp, int i)
{
  for(int j = indexomp; j < (counterVec[i] + indexomp) ;j++)
  {
    // vector<vector<int> > Ctemp;
    // vector<vector<int> > NMinusStemp;
    //vector<vector<int> > Cwhole;
    // generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
    // AllTableElemets[j].NSsubsets=NMinusStemp;
    // AllTableElemets[j].Csubsets=Ctemp;
    //
    // for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j]
    // {
    //     if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
    //     {
    //         AllTableElemets[j].optVector.push_back(0);
    //         break;
    //     }
    //     if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) ( the one that we are doing operation for it ) ----> break (not interested )
    //         break;
    //     for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
    //     {
    //
    //         if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break
    //         {
    //             AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
    //             break;
    //         }
    //     }
    // }
    // int minn = 100000;
    // int myOptimalindex;
    // for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)
    // {
    //     //  cout << AllTableElemets[j].optVector[pp]<<" ";
    //     if(AllTableElemets[j].optVector[pp] < minn)
    //     {
    //         minn=AllTableElemets[j].optVector[pp];
    //         myOptimalindex=pp;
    //
    //     }        gpu_dpFunction<<<0, nthreads0>>>(d_AllGpuTableElements, d_counterVec, indexomp, i);
    // AllTableElemets[j].myOptimalindex=myOptimalindex;
    //
    //
    // if(AllTableElemets[j].NSsubsets.size()>0)
    // {
    //     AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
    // }
  }
}

// void func_name(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp, int i)
// {
//   gpu_dpFunction<<<0, nthreads0>>>(AllGpuTableElements, counterVec, indexomp, i);
// }

void free_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec)
{

}

void init_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec)
{
  int  AllTableElemets_size;
  int  counterVec_size;

  AllTableElemets_size = AllTableElemets.size();
  counterVec_size = counterVec.size();

  h_AllGpuTableElements = new GpuDynamicTable [AllTableElemets_size];
  h_counterVec = new int[counterVec_size];
  h_counterVec = &(counterVec[0]);

  for (int i = 0; i < AllTableElemets_size; i++)
  {
    h_AllGpuTableElements[i].elm = &(AllTableElemets[i].elm[0]);

    for (size_t j = 0; j < AllTableElemets[i].NSsubsets.size(); j++)
      h_AllGpuTableElements[i].NSsubsets[j] = &(AllTableElemets[i].NSsubsets[j][0]);
    for (size_t j = 0; j < AllTableElemets[i].Csubsets.size(); j++)
      h_AllGpuTableElements[i].Csubsets[j] = &(AllTableElemets[i].Csubsets[j][0]);
    for (size_t j = 0; j < AllTableElemets[i].optVector.size(); j++)
      h_AllGpuTableElements[i].optVector.push_back(AllTableElemets[i].optVector[j]);

  	h_AllGpuTableElements[i].myOPT = AllTableElemets[i].myOPT;
    h_AllGpuTableElements[i].mySum = AllTableElemets[i].mySum;
    h_AllGpuTableElements[i].myOptimalindex = AllTableElemets[i].myOptimalindex;
    h_AllGpuTableElements[i].myMinNSVector = &(AllTableElemets[i].myMinNSVector[0]);
  }

  hipMalloc((void**) &d_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size);
  hipMemcpy(d_AllGpuTableElements, h_AllGpuTableElements,
      sizeof(GpuDynamicTable) * AllTableElemets_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_counterVec, sizeof(int) * counterVec_size);
  hipMemcpy(d_counterVec, h_counterVec, sizeof(int) * counterVec_size,
    hipMemcpyHostToDevice);
}
