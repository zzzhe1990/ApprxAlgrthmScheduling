#include "hip/hip_runtime.h"
#include "DPCuda.hh"

GpuDynamicTable *h_AllGpuTableElements;
GpuDynamicTable *d_AllGpuTableElements;
int             AllTableElemets_size;

int             *h_counterVec;
int							*d_counterVec;
int             counterVec_size;

int             *h_roundVec;
int             *d_roundVec;
int             roundVec_size;

__device__
void gpu_sumFun(int *A, int *B , int array_size, int *sum, int d_k)
{
  *sum = 0;

  for (size_t i = 0; i < powf((float)d_k, 2.0); i++)
  {
    *sum = *sum + (A[i] * B[i]);
  }
	// int summ=0.0;
	// for(int i=0; i<(Pow(k,2)); i++)
	// {
	// 	summ= summ + A[i]*B[i];
	// }
	// return summ;
}

__device__
void gpu_increase(const int *Ntemp, int *it, int array_size, bool *result)
{
  for (int i = 0; i < array_size; ++i)
  {
    const int index = array_size - 1 - i;
    ++it[index];
    if (it[index] > Ntemp[index])
    {
      it[index] = 0;
    }
    else {
      *result = true;
      break;
    }
  }
  *result = false;
	// for (int i = 0, size = it.size(); i != size; ++i) {
	// 	const int index = size - 1 - i;
	// 	++it[index];
	// 	if (it[index] > Ntemp[index]) {
	// 		it[index] = 0;
	// 	} else {
	// 		return true;
	// 	}
	// }
	// return false;
}

template<typename T>
__device__ T *gpu_realloc(int oldsize, int newsize, T *old)
{
    T* newT = (T*)malloc(newsize * sizeof(T));

    memcpy(newT, old, oldsize);
    free(old);
    return newT;
}

template<typename T>
__device__ void gpu_push_back(T **array, T *elem, int *size)
{
  (*size)++;
  *array = gpu_realloc<T>((sizeof(T) * (*size - 1)), sizeof(T) * (*size), *array);
	memcpy(*array + (*size - 1), (void*)(elem), sizeof(T));
}

__device__
void gpu_generate2(int *Ntemp, int Ntemp_size, thrust::device_vector<int> *Ctemp, int *Ctemp_Size,
                  thrust::device_vector<int> *NMinusStemp, int d_k, int d_T, int *d_roundVec, thrust::device_vector<int> *tmp_s)
{
  int   *it = new int[Ntemp_size];
  bool  result;
  int   powK;

  powK = powf((float)d_k, 2.0);
  result = true;
  memset(it, 0, Ntemp_size);
  do {
    int *s = new int[Ntemp_size];
    int sSum;

    sSum = 0;
    memset(s, 0, Ntemp_size);
    gpu_sumFun(s, d_roundVec, Ntemp_size, &sSum, d_k);
    if (sSum <= d_T)
    {
      thrust::device_vector<int> *tmp;
      thrust::device_vector<int> *NS;
      int NS_size = 0;

      tmp = (thrust::device_vector<int> *)malloc(sizeof(thrust::device_vector<int>));
      NS = (thrust::device_vector<int> *)malloc(sizeof(thrust::device_vector<int>));
      gpu_push_back<thrust::device_vector<int> >(&Ctemp, tmp, Ctemp_Size);

      for (int j = 0; j < powK; j++)
      {
        // int to_push;
        thrust::device_vector<int> *to_push;

        to_push = (thrust::device_vector<int> *)malloc(sizeof(thrust::device_vector<int>));
        // to_push = ;
        memcpy((void*)&to_push[j],
          (void*)(Ntemp[j] - s[j]),
          sizeof(thrust::device_vector<int>));
        printf("Ici -> %d\n", Ntemp[j] - s[j]);
        // gpu_push_back<thrust::device_vector<int> *>(&NS, &to_push, NS_size);
      }
      // if (NS == Ntemp)
      //   continue;
      //
      // gpu_push_back<thrust::device_vector<int> >(&NMinusStemp, NS, NMinusStemp.size());
      // free (tmp);

      // Ctemp.push_back(s);
      // vector<int> NS;
      // for(int j=0; j<Pow(k,2); j++)
      // {
      //     NS.push_back( Ntemp[j]-s[j]);
      // }
      // if(NS==Ntemp)
      //     continue;
      // NMinusStemp.push_back(NS);
    }

    gpu_increase(Ntemp, it, Ntemp_size, &result);
    free (s);
  } while (result);

  // vector<int> it(Ntemp.size(), 0);
  // do {
  //     vector<int> s;
  //     for(vector<int>::const_iterator i = it.begin(); i != it.end(); ++i)
  //     {
  //         s.push_back(*i);
  //     }
  //     //Cwhole.push_back(s);
  //     int sSum=sumFun(s,roundVec);
  //     if(sSum <= T)
  //     {
  //            Ctemp.push_back(s);
  //
  //         vector<int> NS;
  //         for(int j=0; j<Pow(k,2); j++)
  //         {
  //             NS.push_back( Ntemp[j]-s[j]);
  //         }
  //         if(NS==Ntemp)
  //             continue;
  //         NMinusStemp.push_back(NS);
  //     }
  // }while (increase(Ntemp, it));
}

__global__
void gpu_dpFunction(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp,
                  int i, int d_k, int d_T, int *d_roundVec)
{
  // for(int j = indexomp; j < (counterVec[i] + indexomp) ;j++)
  int threadId = threadIdx.x + indexomp;

  if (threadId < (counterVec[i] + indexomp))
  {
    // thrust::device_vector<int>  *Ctemp;
    // thrust::device_vector<int>  *NMinusStemp;
    // thrust::device_vector<int>  *Cwhole;
    // thrust::device_vector<int>  *tmp_s;
    // int Ctemp_Size = 0;
    // int NMinusStemp_Size = 0;
    // int Cwhole_Size = 0;
    //
    // Ctemp = (thrust::device_vector<int>*)malloc(sizeof(thrust::device_vector<int>*) * Ctemp_Size);
    // NMinusStemp = (thrust::device_vector<int>*)malloc(sizeof(thrust::device_vector<int>*) * NMinusStemp_Size);
    // Cwhole = (thrust::device_vector<int>*)malloc(sizeof(thrust::device_vector<int>*) * Cwhole_Size);
    //
    // gpu_generate2(AllGpuTableElements[threadId].elm, AllGpuTableElements[threadId].elm_size, Ctemp, &Ctemp_Size,
    //   NMinusStemp, d_k, d_T, d_roundVec, tmp_s);
    //
    // AllGpuTableElements[threadId].NSsubsets = NMinusStemp; // not sure if memcpy needed ?
    // AllGpuTableElements[threadId].Csubsets = Ctemp;

    // generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
    // AllTableElemets[j].NSsubsets=NMinusStemp;
    // AllTableElemets[j].Csubsets=Ctemp;
    //
    // for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j]
    // {
    //     if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
    //     {
    //         AllTableElemets[j].optVector.push_back(0);
    //         break;
    //     }
    //     if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) ( the one that we are doing operation for it ) ----> break (not interested )
    //         break;
    //     for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
    //     {
    //
    //         if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break
    //         {
    //             AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
    //             break;
    //         }
    //     }
    // }
    // int minn = 100000;
    // int myOptimalindex;
    // for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)
    // {
    //     //  cout << AllTableElemets[j].optVector[pp]<<" ";
    //     if(AllTableElemets[j].optVector[pp] < minn)
    //     {
    //         minn=AllTableElemets[j].optVector[pp];
    //         myOptimalindex=pp;
    //
    //     }        gpu_dpFunction<<<0, nthreads0>>>(d_AllGpuTableElements, d_counterVec, indexomp, i);
    // AllTableElemets[j].myOptimalindex=myOptimalindex;
    //
    //
    // if(AllTableElemets[j].NSsubsets.size()>0)
    // {
    //     AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
    // }
  }
}

void old_dpFunction(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp, int i)
{
  for(int j = indexomp; j < (counterVec[i] + indexomp) ;j++)
  {
    // vector<vector<int> > Ctemp;
    // vector<vector<int> > NMinusStemp;
    // vector<vector<int> > Cwhole;
    // generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
    // AllTableElemets[j].NSsubsets=NMinusStemp;
    // AllTableElemets[j].Csubsets=Ctemp;
    //
    // for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j]
    // {
    //     if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
    //     {
    //         AllTableElemets[j].optVector.push_back(0);
    //         break;
    //     }
    //     if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) ( the one that we are doing operation for it ) ----> break (not interested )
    //         break;
    //     for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
    //     {
    //
    //         if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break
    //         {
    //             AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
    //             break;
    //         }
    //     }
    // }
    // int minn = 100000;
    // int myOptimalindex;
    // for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)
    // {
    //     //  cout << AllTableElemets[j].optVector[pp]<<" ";
    //     if(AllTableElemets[j].optVector[pp] < minn)
    //     {
    //         minn=AllTableElemets[j].optVector[pp];
    //         myOptimalindex=pp;
    //
    //     }        gpu_dpFunction<<<0, nthreads0>>>(d_AllGpuTableElements, d_counterVec, indexomp, i);
    // AllTableElemets[j].myOptimalindex=myOptimalindex;
    //
    //
    // if(AllTableElemets[j].NSsubsets.size()>0)
    // {
    //     AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
    // }
  }
}

void call_gpu_dpFunction(int indexomp, int i, int k, int T)
{
  gpu_dpFunction<<<1, AllTableElemets_size>>>(
    d_AllGpuTableElements, d_counterVec,
    indexomp, i, k, T, d_roundVec);
}

void free_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec, vector<int> &roundVec)
{
  int tmp;

  hipMemcpy(h_AllGpuTableElements, d_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_counterVec, d_counterVec, sizeof(int) * counterVec_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_roundVec, d_roundVec, sizeof(int) * roundVec.size(), hipMemcpyDeviceToHost);
  hipFree(d_AllGpuTableElements);
  hipFree(d_counterVec);
  hipFree(d_roundVec);

  AllTableElemets.clear(); // destroy elements but don't free the memory
  counterVec.clear();
  roundVec.clear();
  for (int i = 0; i < AllTableElemets_size; i++)
  {
    DynamicTable *tmpTable = new DynamicTable();

    tmpTable->elm.assign(h_AllGpuTableElements[i].elm, h_AllGpuTableElements[i].elm + h_AllGpuTableElements[i].elm_size);

    for (size_t j = 0; j < h_AllGpuTableElements[i].NSsubsets_size; j++) {
      for (size_t k = 0; k < h_AllGpuTableElements[i].NSsubsets[j].size(); k++) {
        tmp = h_AllGpuTableElements[i].NSsubsets[j][k];
        tmpTable->Csubsets[j].push_back(tmp);
      }
    }
    for (size_t j = 0; j < h_AllGpuTableElements[i].Csubsets_size ; j++) {
      for (size_t k = 0; k < h_AllGpuTableElements[j].Csubsets[j].size(); k++) {
        tmp = h_AllGpuTableElements[i].Csubsets[j][k];
        tmpTable->Csubsets[j].push_back(tmp);
      }
    }

    for (size_t j = 0; j < AllTableElemets[i].optVector.size(); j++)
      h_AllGpuTableElements[i].optVector.push_back(AllTableElemets[i].optVector[j]);

    tmpTable->myOPT = h_AllGpuTableElements[i].myOPT;
    tmpTable->mySum = h_AllGpuTableElements[i].mySum;
    tmpTable->myOptimalindex = h_AllGpuTableElements[i].myOptimalindex;
    // h_AllGpuTableElements[i].myMinNSVector = &(AllTableElemets[i].myMinNSVector[0]);

    AllTableElemets.push_back(*tmpTable);
    delete tmpTable;
  }
  counterVec.assign(h_counterVec, h_counterVec + counterVec_size);
  roundVec.assign(h_roundVec, h_roundVec + roundVec_size);
}

void init_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec, vector<int> &roundVec)
{
  AllTableElemets_size = AllTableElemets.size();
  counterVec_size = counterVec.size();
  roundVec_size = roundVec.size();

  h_AllGpuTableElements = new GpuDynamicTable [AllTableElemets_size];
  h_counterVec = new int[counterVec_size];
  h_counterVec = &(counterVec[0]);
  h_roundVec = &(roundVec[0]);

  for (int i = 0; i < AllTableElemets_size; i++)
  {
    h_AllGpuTableElements[i].elm = new int[AllTableElemets[i].elm.size()];
    h_AllGpuTableElements[i].NSsubsets = new thrust::device_vector<int>[AllTableElemets[i].NSsubsets.size()];
    h_AllGpuTableElements[i].Csubsets = new thrust::device_vector<int>[AllTableElemets[i].Csubsets.size()];
    for (size_t j = 0; j < AllTableElemets[i].elm.size(); j++) {
      h_AllGpuTableElements[i].elm[j] = AllTableElemets[i].elm[j];
    }

    for (size_t j = 0; j < AllTableElemets[i].NSsubsets.size(); j++) {
      h_AllGpuTableElements[i].NSsubsets[j] = AllTableElemets[i].NSsubsets[j];
    }
    for (size_t j = 0; j < AllTableElemets[i].Csubsets.size(); j++) {
      h_AllGpuTableElements[i].Csubsets[j] = AllTableElemets[i].Csubsets[j];
    }
    for (size_t j = 0; j < AllTableElemets[i].optVector.size(); j++) {
      h_AllGpuTableElements[i].optVector.push_back(AllTableElemets[i].optVector[j]);
    }

    h_AllGpuTableElements[i].elm_size = AllTableElemets[i].elm.size();
    h_AllGpuTableElements[i].NSsubsets_size = AllTableElemets[i].NSsubsets.size();
    h_AllGpuTableElements[i].Csubsets_size = AllTableElemets[i].Csubsets.size();
  	h_AllGpuTableElements[i].myOPT = AllTableElemets[i].myOPT;
    h_AllGpuTableElements[i].mySum = AllTableElemets[i].mySum;
    h_AllGpuTableElements[i].myOptimalindex = AllTableElemets[i].myOptimalindex;
    h_AllGpuTableElements[i].myMinNSVector = &(AllTableElemets[i].myMinNSVector[0]);
  }

  hipMalloc((void**) &d_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size);
  hipMalloc((void**) &d_counterVec, sizeof(int) * counterVec_size);
  hipMalloc((void**)&d_roundVec, sizeof(int) * roundVec.size());

  hipMemcpy(d_AllGpuTableElements, h_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size, hipMemcpyHostToDevice);
  hipMemcpy(d_counterVec, h_counterVec, sizeof(int) * counterVec_size, hipMemcpyHostToDevice);
  hipMemcpy(d_roundVec, h_roundVec, sizeof(int) * roundVec.size(), hipMemcpyHostToDevice);
}
