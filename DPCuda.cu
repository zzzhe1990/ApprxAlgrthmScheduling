#include "hip/hip_runtime.h"
#include "DPCuda.hh"

GpuDynamicTable *h_AllGpuTableElements;
GpuDynamicTable *d_AllGpuTableElements;
int             *h_counterVec;
int							*d_counterVec;
int             AllTableElemets_size;
int             counterVec_size;

__global__
void gpu_dpFunction(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp, int i)
{
  for(int j = indexomp; j < (counterVec[i] + indexomp) ;j++)
  {
    // vector<vector<int> > Ctemp;
    // vector<vector<int> > NMinusStemp;
    // vector<vector<int> > Cwhole;
    // generate2(AllTableElemets[j].elm,Ctemp,NMinusStemp);
    // AllTableElemets[j].NSsubsets=NMinusStemp;
    // AllTableElemets[j].Csubsets=Ctemp;
    //
    // for(int h=0;h<AllTableElemets[j].NSsubsets.size();h++)   // looking through subset of NSTableElements[j]
    // {
    //     if(AllTableElemets[j].NSsubsets[h]==zeroVec)   // if subset is zero Vector , its OPT is 0
    //     {
    //         AllTableElemets[j].optVector.push_back(0);
    //         break;
    //     }
    //     if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[j].elm)   // if NSsubsets[h] is equal to NSTableElements[j] (itself) ( the one that we are doing operation for it ) ----> break (not interested )
    //         break;
    //     for(int r=0; r<AllTableElemets.size();r++)        // to find the match in the NSTableElements for reaching OPT
    //     {
    //
    //         if(AllTableElemets[j].NSsubsets[h]==AllTableElemets[r].elm)   // if found match of NSsubsets[h], copy its OPT and break
    //         {
    //             AllTableElemets[j].optVector.push_back(AllTableElemets[r].myOPT);
    //             break;
    //         }
    //     }
    // }
    // int minn = 100000;
    // int myOptimalindex;
    // for(int pp=0; pp<AllTableElemets[j].optVector.size();pp++)
    // {
    //     //  cout << AllTableElemets[j].optVector[pp]<<" ";
    //     if(AllTableElemets[j].optVector[pp] < minn)
    //     {
    //         minn=AllTableElemets[j].optVector[pp];
    //         myOptimalindex=pp;
    //
    //     }        gpu_dpFunction<<<0, nthreads0>>>(d_AllGpuTableElements, d_counterVec, indexomp, i);
    // AllTableElemets[j].myOptimalindex=myOptimalindex;
    //
    //
    // if(AllTableElemets[j].NSsubsets.size()>0)
    // {
    //     AllTableElemets[j].myMinNSVector=AllTableElemets[j].NSsubsets[myOptimalindex];
    // }
  }
}

void gpu_generate2(/*vector<int>& Ntemp, vector<vector<int> >& Ctemp, vector<vector<int> >& NMinusStemp*/)
{

}

// void func_name(GpuDynamicTable *AllGpuTableElements, int *counterVec, int indexomp, int i)
// {
//   gpu_dpFunction<<<0, nthreads0>>>(AllGpuTableElements, counterVec, indexomp, i);
// }

/*
  Todo :
    - copy d_AllGpuTableElements to h_AllGpuTableElements - OK
    - copy d_counterVec to h_counterVec - OK
    - free h_AllGpuTableElements - OK
    - free h_counterVec - OK
    - free AllTableElemets ?
    - copy h_AllGpuTableElements to AllTableElemets - KO
    - copy h_counterVec to counterVec - KO
*/

void free_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec)
{
  hipMemcpy(d_AllGpuTableElements, h_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size, hipMemcpyDeviceToHost);
  hipMemcpy(d_counterVec, h_counterVec, sizeof(int) * counterVec_size, hipMemcpyDeviceToHost);
  hipFree(d_AllGpuTableElements);
  hipFree(d_counterVec);
  AllTableElemets.clear(); // destroy elements but don't free the memory
  counterVec.clear();

  for (int i = 0; i < AllTableElemets_size; i++)
  {
    DynamicTable *tmpTable = new DynamicTable();

    tmpTable->elm.assign(h_AllGpuTableElements[i].elm, h_AllGpuTableElements[i].elm + h_AllGpuTableElements[i].elm_size);

    // for (size_t j = 0; j < h_AllGpuTableElements[i].NSsubsets_size; j++) {
    //   for (size_t k = 0; k < h_AllGpuTableElements[i].NSsubsets[j].size(); k++) {
    //     tmpTable->NSsubsets[j].push_back(*h_AllGpuTableElements[i].NSsubsets[j][k].data().get());
    //   }
    // }
    // std::cout << "---------------------------" << '\n';
    int tmp;
    // std::cout << "Csubsets size = " << h_AllGpuTableElements[i].Csubsets_size << '\n';
    for (size_t j = 0; j < h_AllGpuTableElements[i].Csubsets_size ; j++) {
      std::cout << h_AllGpuTableElements[j].Csubsets[j].size() << std::endl;
      for (size_t k = 0; k < h_AllGpuTableElements[j].Csubsets[j].size(); k++) {
        tmp = h_AllGpuTableElements[i].Csubsets[j][k];
        tmpTable->Csubsets[j].push_back(tmp);
        std::cout << "|" << tmpTable->Csubsets[j][k] << ' ';
      }
    }
    std::cout << '\n';

    for (size_t j = 0; j < AllTableElemets[i].optVector.size(); j++)
      h_AllGpuTableElements[i].optVector.push_back(AllTableElemets[i].optVector[j]);

    tmpTable->myOPT = h_AllGpuTableElements[i].myOPT;
    tmpTable->mySum = h_AllGpuTableElements[i].mySum;
    tmpTable->myOptimalindex = h_AllGpuTableElements[i].myOptimalindex;
    // h_AllGpuTableElements[i].myMinNSVector = &(AllTableElemets[i].myMinNSVector[0]);

    AllTableElemets.push_back(*tmpTable);
    delete tmpTable;
  }
  counterVec.assign(h_counterVec, h_counterVec + counterVec_size);
}

void init_gpu(vector<DynamicTable> &AllTableElemets, vector<int> &counterVec)
{
  AllTableElemets_size = AllTableElemets.size();
  counterVec_size = counterVec.size();

  h_AllGpuTableElements = new GpuDynamicTable [AllTableElemets_size];
  h_counterVec = new int[counterVec_size];
  h_counterVec = &(counterVec[0]);

  for (int i = 0; i < AllTableElemets_size; i++)
  {
    h_AllGpuTableElements[i].elm = new int[AllTableElemets[i].elm.size()];
    h_AllGpuTableElements[i].NSsubsets = new thrust::device_vector<int>[AllTableElemets[i].NSsubsets.size()];
    h_AllGpuTableElements[i].Csubsets = new thrust::device_vector<int>[AllTableElemets[i].Csubsets.size()];
    for (size_t j = 0; j < AllTableElemets[i].elm.size(); j++) {
      h_AllGpuTableElements[i].elm[j] = AllTableElemets[i].elm[j];
    }

    for (size_t j = 0; j < AllTableElemets[i].NSsubsets.size(); j++) {
      h_AllGpuTableElements[i].NSsubsets[j] = AllTableElemets[i].NSsubsets[j];
    }
    for (size_t j = 0; j < AllTableElemets[i].Csubsets.size(); j++) {
      h_AllGpuTableElements[i].Csubsets[j] = AllTableElemets[i].Csubsets[j];
    }
    for (size_t j = 0; j < AllTableElemets[i].optVector.size(); j++) {
      h_AllGpuTableElements[i].optVector.push_back(AllTableElemets[i].optVector[j]);
    }

    h_AllGpuTableElements[i].elm_size = AllTableElemets[i].elm.size();
    h_AllGpuTableElements[i].NSsubsets_size = AllTableElemets[i].NSsubsets.size();
    h_AllGpuTableElements[i].Csubsets_size = AllTableElemets[i].Csubsets.size();
  	h_AllGpuTableElements[i].myOPT = AllTableElemets[i].myOPT;
    h_AllGpuTableElements[i].mySum = AllTableElemets[i].mySum;
    h_AllGpuTableElements[i].myOptimalindex = AllTableElemets[i].myOptimalindex;
    h_AllGpuTableElements[i].myMinNSVector = &(AllTableElemets[i].myMinNSVector[0]);
  }

  hipMalloc((void**) &d_AllGpuTableElements, sizeof(GpuDynamicTable) * AllTableElemets_size);
  hipMemcpy(d_AllGpuTableElements, h_AllGpuTableElements,
      sizeof(GpuDynamicTable) * AllTableElemets_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_counterVec, sizeof(int) * counterVec_size);
  hipMemcpy(d_counterVec, h_counterVec, sizeof(int) * counterVec_size,
    hipMemcpyHostToDevice);
}
