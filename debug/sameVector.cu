#include "hip/hip_runtime.h"
#include "sameVector.h"

__device__ int gpu_sameVectors(int *vecA, int *vecB, int size)
{
	int same = 1;
	
	for (int i = 0; i < size; i++)
	{		
		if (vecA[i]!= vecB[i])
/*
		asm(
//			".reg .b64 rd<5>;\n\t"
//			".reg .b32 r<3>;\n\t"
//			".reg .pred %p;\n\t"
//			"ld.param.u64 rd1, %1;\n\t"
//			"ld.param.u64 rd2, %2;\n\t"
//			"move.b32 r1, %3;\n\t"
			"cvt.s64.s32	%%rd0, %3;\n\t"
			"shl.b64 	%%rd0, %%rd0, 2;\n\t"
			"add.s64 	%%rd3, %1, %%rd0;\n\t"
			"add.s64 	%%rd4, %2, %%rd0;\n\t"
			"ld.u32 	%%r2, [%%rd3];\n\t"
			"ld.u32 	%%r3, [%%rd4];\n\t"
			"setp.ne.s32 %%p0, %%r2, %%r3;\n\t"
			"@%%p0 mov.s32 %0, 0;\n\t"
			: "=r"(temp) : "l"(vecA), "l"(vecB), "r"(i)
		);
		if (temp)
*/
		{
			same = 0;
			break;
		}
	}
	return same;
}

__device__ int gpu_sameVectors(int *vecA, int choice, int size)
{
	int vecB[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0};

	vecB[15] = choice;

	int same = 1;
	for (int i = 0; i < size; i++)
	{
		if (vecA[i] != vecB[i])
		{
			same = 0;
			break;
		}
	}
	return same;
}


__global__ void gpu_sameVectors(int *A, int *B, const int powK, int *res)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tRes;
//	__shared__ int warpRes[32];

//	if (thread < 32)
//		warpRes[thread] = 0;

	if (thread < powK)
	{
		tRes = __all(A[thread]-B[thread]);
	}

//	if (thread&(32-1) == 0)
//	{
//		warpRes[thread/32] = tRes;
//	}

//	if (thread < 32)
//		tRes = __any( warpRes != 0 );

	if (thread == 0)
		res[0] = tRes;
}
